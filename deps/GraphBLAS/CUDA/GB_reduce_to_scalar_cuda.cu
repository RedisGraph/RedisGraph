
//------------------------------------------------------------------------------
// GB_reduce_to_scalar_cuda.cu: reduce on the GPU with semiring 
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"

#include "templates/reduceWarp.cu.jit"
#include "templates/reduceNonZombiesWarp.cu.jit"
#include "test/semiringFactory.hpp"

#include "GB_jit_launcher.h"
#include "GB_callback.hpp"


const std::vector<std::string> header_names ={};

GrB_Info GB_reduce_to_scalar_cuda
(
    GB_void *s,
    const GrB_Monoid reduce,
    const GrB_Matrix A,
    GB_Context Context
)
{ 

    printf ("Hi I am %s :-)\n", __FILE__) ;

    // result = sum (Anz [0..anz-1]) using the GPU,
    // with a kernel that has ntasks = grid.x and blocksize = blockDim.x
    // nthreads = # of GPUs to use, but 1 for now
    // We have a workspace W of size ntasks.

    thread_local static jitify::JitCache kernel_cache;
    std::string reduce_kernel_name = "reduceNonZombiesWarp";

    // stringified kernel specified above
    jitify::Program program= kernel_cache.program( templates_reduceNonZombiesWarp_cu, 0, 0,
        file_callback_plus);
    //{"--use_fast_math", "-I/usr/local/cuda/include"});

    int nnz = GB_NNZ( A ) ;
    GrB_Type ctype = reduce->op->ztype ;

    int blocksize = 1024 ;
    int ntasks = ( nnz + blocksize -1) / blocksize ;

    int32_t *block_sum;
    //hipMallocManaged ((void**) &block_sum, (num_reduce_blocks)*sizeof(int32_t)) ;
    block_sum = (int32_t*)GB_cuda_malloc( (ntasks)*sizeof(int32_t)) ;

    dim3 red_grid(ntasks);
    dim3 red_block(blocksize);

    GBURBLE ("(GPU reduce launch nblocks,blocksize= %d,%d )\n", ntasks, blocksize) ;
    jit::launcher( reduce_kernel_name + "_" + reduce->op->name,
                   templates_reduceNonZombiesWarp_cu,
                   header_names,
                   compiler_flags,
                   callback_wrapper)
                   .set_kernel_inst( reduce_kernel_name , { ctype->name })
                   .configure(red_grid, red_block) //if commented, use implicit 1D configure in launch
                   .launch(
                            A->i,   // index vector, only sum up values >= 0
                            A->x,   // input pointer to vector to reduce, with zombies
                            block_sum,             // Block sums on return 
                            (unsigned int)nnz      // length of vector to reduce to scalar

                        );

    hipDeviceSynchronize();


    for (int i = 0 ; i < ntasks ; i++)
    {
        *s += (block_sum [i]) ; 
    }


    return (GrB_SUCCESS) ;
}

