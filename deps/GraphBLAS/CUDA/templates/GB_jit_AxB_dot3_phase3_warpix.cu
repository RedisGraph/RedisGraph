#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// AxB_dot3_phase3_warpix.cu 
//------------------------------------------------------------------------------

// This CUDA kernel produces the semi-ring product of two
// sparse matrices of types T_A and T_B and common index space size n, to a  
// output matrix of type T_C. The matrices are sparse, with different numbers
// of non-zeros and different sparsity patterns. 
// ie. we want to produce C = A'*B in the sense of the given semi-ring.

// This version uses a merge-path algorithm, when the sizes nnzA and nnzB are 
// relatively close in size, neither is very spare nor dense, for any size of N.
// Handles arbitrary sparsity patterns with guaranteed load balance.

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x. s= 32 with a variable number
// of active threads = min( min(g_xnz, g_ynz), 32) 

// Thus, threadblock b owns a part of the index set spanned by g_xi and g_yi.  Its job
// is to find the intersection of the index sets g_xi and g_yi, perform the semi-ring dot
// product on those items in the intersection, and finally reduce this data to a scalar, 
// on exit write it to g_odata [b].

//  int64_t start          <- start of vector pairs for this kernel
//  int64_t end            <- end of vector pairs for this kernel
//  int64_t *Bucket        <- array of pair indices for all kernels 
//  matrix<T_C> *C         <- result matrix 
//  matrix<T_M> *M         <- mask matrix
//  matrix<T_A> *A         <- input matrix A
//  matrix<T_B> *B         <- input matrix B
#define GB_KERNEL
#include <limits>
#include <cstdint>
#include "matrix.h"
#include <hip/hip_cooperative_groups.h>
#include "mySemiRing.h"

// Using tile size fixed at compile time, we don't need shared memory
#define tile_sz 32 

using namespace cooperative_groups;

template< typename T, int warp_sz>
__device__ __inline__ 
T GB_reduce_sum(thread_block_tile<warp_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        T next = g.shfl_down( val, i);
        val = GB_ADD( val, next ) ;
    }
    return val;
}

template< typename T, int warp_sz>
__device__ __inline__ 
T reduce_plus(thread_block_tile<warp_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        val += g.shfl_down( val, i) ;
    }
    return val; // note: only thread 0 will return full sum and flag value
}

#define intersects_per_thread 8

template< typename T_C, typename T_A, typename T_B, typename T_X, typename T_Y, typename T_Z>  
__global__ void AxB_dot3_phase3_warpix
(
    int64_t start,
    int64_t end,
    int64_t *__restrict__ Bucket,
    GrB_Matrix C,
    GrB_Matrix M,
    GrB_Matrix A,
    GrB_Matrix B,
    int sz
)
{

    T_A *__restrict__ Ax = (T_A*)A->x;
    T_B *__restrict__ Bx = (T_B*)B->x;
    T_C *__restrict__ Cx = (T_C*)C->x;
    int64_t *__restrict__ Ci = C->i;
    int64_t *__restrict__ Mi = M->i;
    int64_t *__restrict__ Mp = M->p;
    int64_t *__restrict__ Ai = A->i;
    int64_t *__restrict__ Bi = B->i;
    int64_t *__restrict__ Ap = A->p;
    int64_t *__restrict__ Bp = B->p;

    int64_t mnvec = M->nvec;

    // zombie count
    int zc;

    int64_t pair_id;

    // set thread ID
    int tid_global = threadIdx.x+ blockDim.x* blockIdx.x;
    int tid = threadIdx.x;
    int b = blockIdx.x ;

    // total items to be inspected
    int64_t nnzA = 0;
    int64_t nnzB = 0;

    thread_block_tile<tile_sz> tile = tiled_partition<tile_sz>( this_thread_block());

    //int parts = gridDim.x; //Each warp is a part

    //Find our part of the work bucket
    int64_t pfirst, plast, kfirst, klast ;
    GB_PARTITION (pfirst, plast, end-start, b, gridDim.x ) ;
    /* 
    if( tid ==0 ) {
       printf("block%d is alive, pf,pl=%ld,%ld \n", b, pfirst, plast);
    }
    __syncthreads();
    */
    
    
    __shared__ int64_t As[256];
    __shared__ int64_t Bs[256];
    __shared__ T_A Axs[256]; 
    __shared__ T_B Bxs[256]; 

   /* 
    int Bpl[9]; // local offsets into shared for multiple vectors of B
    int shr_vec[8] ; //columns of B we see in this task

    pair_id = Bucket[pfirst];
    int64_t i = Mi[pair_id] ;
    int vecs = 1 ;
    int last_vec = i;
    shr_vec[0] = i;
    for (int id =1; id< plast-pfirst; id++)
    {
         pair_id = Bucket[pfirst+id];
         i = Mi[pair_id];
         if (i == last_vec) continue;
         vecs++;
         shr_vec[vecs] = i;
         last_vec = i;
    }
    int all_loaded = 0;

    Bpl[0] = 0;
    for ( int k = 0; k < vecs; k++)
    {   
        int64_t pA       = Ap[ shr_vec[k] ]; 
        int64_t pA_end   = Ap[ shr_vec[k] +1]; 
        nnzA = pA_end - pA;
        Bpl[k+1] = Bpl[k] + nnzA;
        for (int i = tid ; i < nnzA; i+= blockDim.x)
        {
           As[ Bpl[k] +i ] = Ai[ pA + i ] ; 
        }
        __syncthreads();
    }

    //pre-load columns of B, which will be reused, to shared memory
    //Due to loading a contigious block with stride 1 this is fast
        
    all_loaded = (Bpl[vecs] < 256 );
    if( tid == 0 ) {
       printf("block%d loaded %d vals from B, vecs=%d, all_loaded=%d\n",
                 b, Bpl[vecs], vecs, all_loaded );
    }
    __syncthreads();


    // reset counter
    */
    // Main loop over pairs 
    for (int id = start + pfirst; // loop on pairs 
         id < start+ plast;  
         id ++ )
    {
         int64_t pair_id = Bucket[id];
          
         int64_t i = Mi[pair_id];
         int64_t j = Ci[pair_id] >> 4;

         int64_t pA       = Ap[i];
         int64_t pA_end   = Ap[i+1];
         nnzA = pA_end - pA;

         int64_t pB       = Bp[j]; 
         int64_t pB_end   = Bp[j+1]; 
         nnzB = pB_end - pB;

         zc = 0 ;
         int j_last = -1 ;
         
         
    // No search, this warp does all the work

    int tx_start = pA;
    int tx_end   = pA_end;
    int ty_start = pB;
    int ty_end   = pB_end;

    for ( int i = tid; i < nnzA ; i+= blockDim.x)
    {
       As [i] = Ai[ pA + i];
       Axs[i] = Ax[ pA + i];
    }
    __syncthreads();

    if ( j != j_last) { 
        for ( int i = tid; i < nnzB ; i+= blockDim.x)
        {
           Bs [i] = Bi[ pB + i];
           Bxs[i] = Bx[ pB + i];
        }
        __syncthreads();
        j_last = j;
    }
    

    /*     
    if ( tid==0 ) {
      //printf("block %d dot %lld i,j= %lld,%lld\n", blockIdx.x, pair_id, i, j);
      printf("block%d dot %ld(i,j)=(%ld,%ld) xs,xe= %d,%d ys,ye = %d,%d \n", 
               b, pair_id, i, j, tx_start,tx_end, ty_start, ty_end);
      //for(int a = 0; a < nnzA; a++) printf(" As[%d]:%ld ",a, As[j]);
    }
    tile.sync();
    */
    
    

    // Warp intersection: balanced by design, no idle threads. 
    // Each 32 thread warp will handle 32 comparisons per loop.
    // Either A or B takes stride 4, other takes stride 8
    // For this version A strides 4, B strides 8
    T_A aki;
    T_B bkj;
    T_Z cij = GB_IDENTITY ;
    int Astride = nnzA > nnzB ? 8 : 4;
    int Ashift  = nnzA > nnzB ? 3 : 2;
    int Amask   = nnzA > nnzB ? 7 : 3;
    int Bstride = nnzB >= nnzA ? 8 : 4;
    //printf(" Astride = %d, Bstride = %d\n", Astride, Bstride);

    // TODO PLUS_PAIR_INT64, FP32, FP64: no need for cij_exists.
    // just check if cij > 0

    int cij_exists  = 0 ;

    //Warp intersection dot product
    int bitty_row = tid &  Amask ;
    int bitty_col = tid >> Ashift ;

    int k = tx_start + bitty_row ;
    int l = ty_start + bitty_col ;

    //Ai[k] = As[ k -pA ];  for lookup
    //Bi[l] = Bs[ l -pB ]; 


    int inc_k,inc_l;

    int active = ( ( k < tx_end) && (l < ty_end ) );
       
    /*    
    printf("block%d tid%d  Ai,As=%ld,%ld Bi,Bs=%ld,%ld  k,l =%d,%d active:%d\n",
                    b,tid, Ai[k], As[k -pA], Bi[l], Bs[l -pB],
                    k, l,  active );
    */
                    
    
    while ( tile.any(active) )
    {
       inc_k = 0;
       inc_l = 0;
       int kp = k-pA;
       int lp = l-pB;
       if ( active )
       { 
          coalesced_group g = coalesced_threads();
          if ( g.thread_rank() == g.size()-1)
          {
             inc_k = ( As[kp] <= Bs[lp] ) ;
             inc_l = ( Bs[lp] <= As[kp] ) ;
             // printf("block%d tid%d inc_k= %d inc_l = %d\n",b, tid, inc_k, inc_l );
          }
          //tile.sync();

          if ( As [kp] == Bs [lp] )
          {
              //Axs[kp] = Ax[k];
              //Bxs[lp] = Bx[l];

              GB_GETA ( aki=(T_Z)Axs[kp] ) ;
              GB_GETB ( bkj=(T_Z)Bxs[lp] ) ;
              if (cij_exists)
              {
                T_Z t = GB_MULT( (T_Z) aki, (T_Z) bkj);
                GB_ADD_F( cij, t ) ;
                //printf("block%d  thd%d ix at %ld(%ld)  cij += %d * %d\n",b, tid, Ai[k], As[kp], aki, bkj);
              }
              else
              {
                cij_exists = 1 ;
                cij = GB_MULT ( (T_Z) aki, (T_Z) bkj) ;
                //printf("  thd%d ix at %ld(%ld)  cij = %d * %d \n", tid, Ai[k], Ais[kp], aki, bkj);
              }
          }
          // TODO check terminal condition
          //printf(" block%u work value = %d, exists = %d\n", b, cij, cij_exists);
          //printf("block%d tid%d k,l = %d,%d Ai,Bi = %ld,%ld \n", b, tid, k, l, Ai[k], Bi[l] );
       }
       //tile.sync();
       //inc_k = tile.shfl_down( inc_k, 31-tid);
       if( tile.any(inc_k) ) {
          k =1+ tile.shfl_down(k,31-tid) + bitty_row ; // tid%Astride;
          //Ais [k-pA] = As[k-pA];
          //Axs [bitty_row] = Ax[k];
       }
       if( tile.any(inc_l) ) {
          l =1+ tile.shfl_down(l,31-tid) + bitty_col ; // tid/Astride;
          //Bis [l-pB] = Bs[l-pB];
          //Bxs [bitty_col] = Bx[l];
       }
       active = ( ( k < tx_end) && (l < ty_end ) );
       //printf("block%d tid = %d k = %d l= %d active=%d\n", b, tid, k, l,active);
    }
    tile.sync();

    //--------------------------------------------------------------------------
    // reduce sum per-thread values to a single scalar, get OR of flag
    //--------------------------------------------------------------------------

    // Do vote here for control.
    cij_exists  = tile.any( cij_exists);
    tile.sync();

    if (cij_exists)
    {
       cij = GB_reduce_sum<T_Z, tile_sz>( tile, cij );
    }
    tile.sync();
    

    // Atomic write result for this block to global mem
    if (tid == 0)
    {
        //printf ("final %d : %d exists = %d\n", b,  cij, cij_exists) ;
        if (cij_exists)
        {
           //printf("block%d i,j =%ld,%ld cij = %d\n",b, i, j, cij);
           GB_PUTC( Cx[pair_id] = (T_C) cij ) ;
           GB_PUTC ( Ci[pair_id] = i ) ;
           
        }
        else
        {
            //printf(" dot %d is a zombie\n", pair_id);
            zc++;
            GB_PUTC ( Ci[pair_id] = GB_FLIP (i) ) ;
        }
    
    //__syncthreads(); 
  

       if( zc > 0)
       {
          //printf("warp %d zombie count = %d\n", blockIdx.x, zc);
          atomicAdd( (unsigned long long int*)&(C->nzombies), (unsigned long long int)zc);
          //printf("blk:%d Czombie = %lld\n",blockIdx.x,C->zombies);
       }

    }
    tile.sync();
    /*
    */
  }
}

