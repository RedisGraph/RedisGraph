#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// templates/GB_AxB_cuda_dot3_phase2: fill the global buckets
//------------------------------------------------------------------------------

// TODO describe me

#define GB_KERNEL
#include <cstdint>
#include "GB_cuda_buckets.h"
#include "matrix.h"
#include <hip/hip_cooperative_groups.h>
#include "local_cub/block/block_scan.cuh"

using namespace cooperative_groups;

// A stateful callback functor that maintains a running prefix to be applied
// during consecutive scan operations.
struct BlockPrefixCallbackOp
{
   // Running prefix
   int64_t running_total;
   // Constructor
   __device__ BlockPrefixCallbackOp(int64_t running_total) : running_total(running_total) {}

   // Callback operator to be entered by the first warp of threads in the block.
   // Thread-0 is responsible for returning a value for seeding the block-wide scan.
   __device__ int64_t operator()(int64_t block_aggregate)
   {
     int64_t old_prefix = running_total;
     running_total += block_aggregate;
     return old_prefix;
   }
};

__inline__ 
__device__ void blockBucketExclusiveSum(int bucketId, int64_t *d_data, int nblocks)
{
   #define blocksize  32

   // Specialize BlockScan for a 1D block of 32 threads
   typedef hipcub::BlockScan<int64_t, 32, hipcub::BLOCK_SCAN_WARP_SCANS> BlockScan; 

   // Allocate shared memory for BlockScan
   __shared__ typename BlockScan::TempStorage temp_storage;

   // Initialize running total
   BlockPrefixCallbackOp prefix_op(0);

   // Have the block iterate over segments of items
   int64_t data=0;

   int64_t *blockbucket= d_data;

   for (int block_id = 0; block_id < nblocks; block_id += blocksize)
   {
    // Load a segment of consecutive items that are blocked across threads

    //printf("block %d entering sum\n",blockIdx.x);
      int loc = block_id + threadIdx.x;
      if ( loc < nblocks)
      { 
        //printf("block %di loading tid=%d\n",block_id,tid);
        data  = blockbucket[bucketId*nblocks    +loc ] ; 
      }
      __syncthreads();

      //printf("bb%d_%d s0 before prefix= %ld \n", block_id,bucketId, 
      //                     blockbucket[bucketId*nblocks + block_id+threadIdx.x] )  ; 
      // Collectively compute the block-wide exclusive prefix sum
      BlockScan(temp_storage).ExclusiveSum( data, data, prefix_op);
      __syncthreads();

      if ( loc < nblocks)
      { 
        blockbucket[bucketId*nblocks   +loc ]  = data  ; 
      }
      __syncthreads();

        //printf("bb%d_%d = %ld \n", block_id, bucketId, blockbucket[bucketId*nblocks+block_id+threadIdx.x] )  ; 
      
      data = 0;
   }
}


template< typename T, int tile_sz>
__inline__ __device__ 
T warp_ReduceSumPlus( thread_block_tile<tile_sz> tile, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = tile.size() / 2; i > 0; i /= 2) {
        val +=  tile.shfl_down( val, i);
    }
    return val; // note: only thread 0 will return full sum
}

template<typename T, int warpSize>
__inline__ __device__
T block_ReduceSum(thread_block g, T val)
{
  static __shared__ T shared[warpSize]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;
  thread_block_tile<warpSize> tile = tiled_partition<warpSize>( g );

  // Each warp performs partial reduction
  val = warp_ReduceSumPlus<T, warpSize>( tile, val);    

  // Wait for all partial reductions
  if (lane==0) { 
     //printf("thd%d warp%d sum is %d\n", threadIdx.x, wid, val);
     shared[wid]=val; // Write reduced value to shared memory
     //printf("thd%d stored warp %d sum %d\n", threadIdx.x, wid, val);
  }
  __syncthreads();              // Wait for all partial reductions

  if (wid > 0 ) return val ;
  //Final reduce within first warp
  if (wid==0) val = warp_ReduceSumPlus<T, warpSize>( tile, val) ; 

  return val;
}

// GB_AxB_cuda_dot3_phase2 is a CUDA kernel that takes as input the
// nanobuckets and blockbucket arrays computed by the first phase kernel,
// GB_AxB_cuda_dot3_phase1.  The launch geometry of this kernel must match the
// GB_AxB_cuda_dot3_phase1 kernel, with the same # of threads and threadblocks.

__global__ 
void GB_AxB_dot3_phase2
(
    // input, not modified:
    int64_t *__restrict__ nanobuckets,    // array of size 12-blockDim.x-by-nblocks
    int64_t *__restrict__ blockbucket,    // global bucket count, of size 12*nblocks
    // output:
    int64_t *__restrict__ bucketp,        // global bucket cumsum, of size 13 
    int64_t *__restrict__ bucket,         // global buckets, of size cnz (== mnz)
    int64_t *__restrict__ offset,         // global offsets, for each bucket
    // inputs, not modified:
    GrB_Matrix C,             // output matrix
    const int64_t cnz,        // number of entries in C and M 
    const int nblocks         // input number of blocks to reduce
)
{

    //--------------------------------------------------------------------------
    // get C and M
    //--------------------------------------------------------------------------

    //int64_t *Ci = C->i ;       // for zombies, or bucket assignment

    // Ci [p] for an entry C(i,j) contains either GB_FLIP(i) if C(i,j) is a
    // zombie, or (k << 4) + bucket otherwise, where C(:,j) is the kth vector
    // of C (j = Ch [k] if hypersparse or j = k if standard sparse), and
    // where bucket is the bucket assignment for C(i,j).  This phase does not
    // need k, just the bucket for each entry C(i,j).

    //--------------------------------------------------------------------------
    // sum up the bucket counts of prior threadblocks
    //--------------------------------------------------------------------------

    // blockbucket is an array of size 12-by-nblocks, held by row.  The
    // entry blockbucket [bucket * nblocks + t] holds the # of entries
    // in the bucket (in range 0 to 11) found by threadblock t.


    //__shared__ uint64_t offset [12] ;
    uint64_t s_0=0;
    uint64_t s_1=0;
    uint64_t s_2=0;
    uint64_t s_3=0;
    uint64_t s_4=0;
    uint64_t s_5=0;
    uint64_t s_6=0;
    uint64_t s_7=0;
    uint64_t s_8=0;
    uint64_t s_9=0;
    uint64_t s_10=0;
    uint64_t s_11=0;

    thread_block_tile<32> tile = tiled_partition<32>(this_thread_block() );

    //printf("block %d entering sum\n",blockIdx.x);
    int tid = threadIdx.x  + blockIdx.x*blockDim.x;
    #define reduceBucket( B )    \
     for( tid = threadIdx.x + blockIdx.x*blockDim.x; \
          tid < nblocks;  \
          tid += blockDim.x*gridDim.x) \
     {                           \
        s_ ## B  += blockbucket[  B *nblocks +tid] ;  \
     } \
     __syncthreads(); \
     s_ ## B  = warp_ReduceSumPlus<uint64_t , 32>( tile, s_ ## B); 

     reduceBucket( 0 )
     reduceBucket( 1 )
     reduceBucket( 2 )
     reduceBucket( 3 )
     reduceBucket( 4 )
     reduceBucket( 5 )
     reduceBucket( 6 )
     reduceBucket( 7 )
     reduceBucket( 8 )
     reduceBucket( 9 )
     reduceBucket( 10 )
     reduceBucket( 11 )


        //printf("summing blk,tid=%d,%d\n",blockIdx.x,threadIdx.x);
       if (threadIdx.x ==0 )
       {
          atomicAdd( (unsigned long long int*)&(offset[0]), s_0);
          atomicAdd( (unsigned long long int*)&(offset[1]), s_1);
          atomicAdd( (unsigned long long int*)&(offset[2]), s_2);
          atomicAdd( (unsigned long long int*)&(offset[3]), s_3);
          atomicAdd( (unsigned long long int*)&(offset[4]), s_4);
          atomicAdd( (unsigned long long int*)&(offset[5]), s_5);
          atomicAdd( (unsigned long long int*)&(offset[6]), s_6);
          atomicAdd( (unsigned long long int*)&(offset[7]), s_7);
          atomicAdd( (unsigned long long int*)&(offset[8]), s_8);
          atomicAdd( (unsigned long long int*)&(offset[9]), s_9);
          atomicAdd( (unsigned long long int*)&(offset[10]),s_10);
          atomicAdd( (unsigned long long int*)&(offset[11]),s_11);
       }
       __syncthreads();
       


    if( gridDim.x >= 12)
    {
        // Cumulative sum across blocks for each bucket 
        if (blockIdx.x <12)
           blockBucketExclusiveSum( blockIdx.x, blockbucket, nblocks ) ;
    }
    else
    {
        if (blockIdx.x == 0)
        {
           blockBucketExclusiveSum( 0, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 1, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 2, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 3, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 4, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 5, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 6, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 7, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 8, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 9, blockbucket, nblocks ) ;
           blockBucketExclusiveSum( 10, blockbucket, nblocks) ;
           blockBucketExclusiveSum( 11, blockbucket, nblocks) ;
        }
    }
    
    
    

    //--------------------------------------------------------------------------
    // last threadblock saves the cumsum of the 12 global buckets
    //--------------------------------------------------------------------------
    /* do on cpu
    if (blockIdx.x == 0) // gridDim.x - 1)
    {

        // the last threadblock: compute all 12 global bucket sizes, and its
        // cumulative sum
        if (threadIdx.x == 0)
        {
            // the work in this last threadblock is single-threaded
            uint64_t s = 0;
            for (int bucket = 0 ; bucket < 12 ; bucket++)
            {
                // write the global cumsum of all buckets to the final global
                // bucketp.  bucketp [bucket] is the starting position in
                // the bucket.
                bucketp [bucket] = s ;
                
                // bucket_size is the total # of entries in this bucket, for
                // all threadblocks.  It has nearly been computed already,
                // since offset [bucket] = sum (blockbucket (bucket,0:blockDim.x-1)).
                // All that is left is to add the counts for the last threadblock.`
                //int64_t global_bucket_size = offset [bucket];   
                     // + blockbucket [bucket * gridDim.x + blockIdx.x] ;

                //printf("bucketp[%d]= %ld\n",bucket, s);
                // s is a cumulative sum of the global bucket sizes
                s += offset[bucket]; // global_bucket_size ;
            }
            // The kth global bucket (for k = 0 to 11) appears in:
            // bucket [bucketp [k]... bucketp [k+1]-1],
            // so the end of the last bucket needs bucketp [12].
            bucketp [12] = (int64_t)s;
                //printf("bucketp[12]= %ld\n", s);
            // all entries in C now appear in the buckets.
            // ASSERT (s == cnz) ;
        }
        __syncthreads ( ) ;
    }
    */

} // phase2 


__global__ 
void GB_AxB_dot3_phase2end
(
    // input, not modified:
    int64_t *__restrict__ nanobuckets,    // array of size 12-blockDim.x-by-nblocks
    const int64_t *__restrict__ blockbucket,    // global bucket count, of size 12*nblocks
    // output:
    const int64_t *__restrict__ bucketp,        // global bucket cumsum, of size 13 
    int64_t *__restrict__ bucket,         // global buckets, of size cnz (== mnz)
    const int64_t *__restrict__ offset,        // global offsets, for each bucket
    // inputs, not modified:
    const GrB_Matrix C,            // output matrix
    const int64_t cnz        // number of entries in C and M 
)
{


    int64_t *__restrict__ Ci = C->i ;       // for zombies, or bucket assignment
    int64_t *__restrict__ Mp = C->p ;       // for offset calculations 
    int64_t mnvec = C->nvec;

    //--------------------------------------------------------------------------
    // load and shift the nanobuckets for this thread block
    //--------------------------------------------------------------------------

    // The taskbucket for this threadblock is an array of size
    // 12-by-blockDim.x, held by row.  It forms a 2D array within the 3D
    // nanobuckets array.
    int64_t *__restrict__ taskbucket = nanobuckets + blockIdx.x * (12 * blockDim.x) ;

    //printf("block%d thd%d blockbucket= %ld\n", blockIdx.x, threadIdx.x, 
    //                                           blockbucket[blockIdx.x*gridDim.x+blockIdx.x]);

    // Each thread in this threadblock owns one column of this taskbucket, for
    // its set of 12 nanobuckets.  The nanobuckets are a column of length 12,
    // with stride equal to blockDim.x.
    int64_t *__restrict__ nanobucket = taskbucket + threadIdx.x;

    // Each thread loads its 12 nanobucket values into registers.
    #define LOAD_NANOBUCKET(bucket)                     \
        int64_t my_bucket_ ## bucket =                  \
            nanobucket [bucket * blockDim.x]            \
         + blockbucket [bucket * gridDim.x + blockIdx.x]\
         + bucketp [bucket] ;                          

    LOAD_NANOBUCKET (0) ;
    LOAD_NANOBUCKET (1) ;
    LOAD_NANOBUCKET (2) ;
    LOAD_NANOBUCKET (3) ;
    LOAD_NANOBUCKET (4) ;
    LOAD_NANOBUCKET (5) ;
    LOAD_NANOBUCKET (6) ;
    LOAD_NANOBUCKET (7) ;
    LOAD_NANOBUCKET (8) ;
    LOAD_NANOBUCKET (9) ;
    LOAD_NANOBUCKET (10) ;
    LOAD_NANOBUCKET (11) ;

    // Now each thread has an index into the global set of 12 buckets,
    // held in bucket, of where to place its own entries.

    //--------------------------------------------------------------------------
    // construct the global buckets
    //--------------------------------------------------------------------------

    // The slice for task blockIdx.x contains entries pfirst:plast-1 of M and
    // C, which is the part of C operated on by this threadblock.
    int64_t pfirst, plast ;

    /*
    for ( int tid_global = threadIdx.x + blockIdx.x * blockDim.x ;
              tid_global < (mnvec+7)/8 ;
              tid_global += blockDim.x * gridDim.x)
    */
    int chunk_max= (cnz + chunksize -1)/chunksize;
    for ( int chunk = blockIdx.x;
              chunk < chunk_max;
              chunk += gridDim.x ) 
    {

    //GB_PARTITION (pfirst, plast, cnz, tid_global, (mnvec+7)/8 ) ;
      pfirst = chunksize * chunk ; 
      plast  = GB_IMIN( chunksize * (chunk+1), cnz ) ;

      int chunk_end;
      if ( cnz > chunksize) chunk_end = GB_IMIN(  chunksize, 
                                                  cnz - chunksize*(chunk) ); 
      else chunk_end = cnz;

    // find the first vector of the slice for task blockIdx.x: the
    // vector that owns the entry Ai [pfirst] and Ax [pfirst].
    //kfirst = GB_search_for_vector_device (pfirst, Mp, 0, mnvec) ;

    // find the last vector of the slice for task blockIdx.x: the
    // vector that owns the entry Ai [plast-1] and Ax [plast-1].
    //klast = GB_search_for_vector_device (plast-1, Mp, kfirst, mnvec) ;
    

    for ( int p = pfirst + threadIdx.x;
              p < pfirst + chunk_end;
              p += blockDim.x )
    {
        // get the entry C(i,j), and extract its bucket.  Then
        // place the entry C(i,j) in the global bucket it belongs to.

        // TODO: these writes to global are not coalesced.  Instead: each
        // threadblock could buffer its writes to 12 buffers and when the
        // buffers are full they can be written to global.
        int ibucket = Ci[p] & 0xF;
        //printf(" thd: %d p,Ci[p] = %ld,%ld,%d\n", threadIdx.x, p, Ci[p], irow );
        switch (ibucket)
        {
            case  0: bucket [my_bucket_0++ ] = p ; Ci[p] = Ci[p] >>4; break ; //unshift zombies
            case  1: bucket [my_bucket_1++ ] = p ; break ;
            case  2: bucket [my_bucket_2++ ] = p ; break ;
            case  3: bucket [my_bucket_3++ ] = p ; break ;
            case  4: bucket [my_bucket_4++ ] = p ; break ;
            case  5: bucket [my_bucket_5++ ] = p ; break ;
            case  6: bucket [my_bucket_6++ ] = p ; break ;
            case  7: bucket [my_bucket_7++ ] = p ; break ;
            case  8: bucket [my_bucket_8++ ] = p ; break ;
            case  9: bucket [my_bucket_9++ ] = p ; break ;
            case 10: bucket [my_bucket_10++] = p ; break ;
            case 11: bucket [my_bucket_11++] = p ; break ;
            default: break; 
        }
        
    }
    //__syncthreads();
  } 
    
}

