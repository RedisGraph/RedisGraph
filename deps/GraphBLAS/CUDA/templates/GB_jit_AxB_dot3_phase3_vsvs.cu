#include "hip/hip_runtime.h"
//******************************************************************************
//  Sparse dot version of Matrix-Matrix multiply with mask 
//  Each thread in this kernel is responsible for m vector-pairs(x,y), 
//  finding intersections and producting the final dot product for each
//  using a serial merge algorithm on the sparse vectors. 
//  m = 256/sz, where sz is in {4, 16, 64, 256}
//  For a vector-pair, sz = xnz + ynz 
//  Template on <T_C, T_M, T_A, T_B>
//  Parameters:

//  int64_t start          <- start of vector pairs for this kernel
//  int64_t end            <- end of vector pairs for this kernel
//  int64_t *Bucket        <- array of pair indices for all kernels 
//  matrix<T_C> *C         <- result matrix 
//  matrix<T_M> *M         <- mask matrix
//  matrix<T_A> *A         <- input matrix A
//  matrix<T_B> *B         <- input matrix B
//  int sz                 <- nnz of very sparse vectors

//  Blocksize is 1024, uses warp and block reductions to count zombies produced.
//******************************************************************************
#define GB_KERNEL
#include <limits>
#include <cstdint>
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include "matrix.h"
#include "mySemiRing.h"

using namespace cooperative_groups;

template< typename T, int tile_sz>
__inline__ __device__ 
T warp_ReduceSumPlus( thread_block_tile<tile_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        //printf("thd%d   %d OP %d is %d\n", threadIdx.x, val, fold, OP( val, fold));
        val +=  g.shfl_down( val, i);
    }
    return val; // note: only thread 0 will return full sum
}

template< typename T, int tile_sz>
__inline__ __device__ 
T warp_Reduce( thread_block_tile<tile_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        T next = g.shfl_down( val, i) ;
        val = GB_ADD( sum, next ) ; 
    }
    return val; // note: only thread 0 will return full sum
}

template<typename T, int warpSize>
__inline__ __device__
T block_ReduceSum(thread_block g, T val)
{
  static __shared__ T shared[warpSize]; // Shared mem for 32 partial sums
  int lane = threadIdx.x & 31 ; // % warpSize;
  int wid  = threadIdx.x >> 5 ; // / warpSize;
  thread_block_tile<warpSize> tile = tiled_partition<warpSize>( g );

  // Each warp performs partial reduction
  val = warp_ReduceSumPlus<T, warpSize>( tile, val);    

  // Wait for all partial reductions
  if (lane==0) shared[wid]=val; // Write reduced value to shared memory
  __syncthreads();              // Wait for all partial reductions

  if (wid > 0 || gridDim.x == 1 ) return val;

  //read from shared memory only if that warp existed
  val = (threadIdx.x <  (blockDim.x / warpSize ) ) ? shared[lane] : 0;
  //printf("thd%d warp loaded val = %d\n", threadIdx.x, lane, val);

  if (wid==0) val = warp_ReduceSumPlus<T, warpSize>( tile, val); //Final reduce within first warp

  return val;
}

template< typename T_C, typename T_A, typename T_B, typename T_X, typename T_Y, typename T_Z>
__global__ void AxB_dot3_phase3_vsvs
( 
  const int64_t start, 
  const int64_t end,
  const int64_t *__restrict__ Bucket, 
  const GrB_Matrix C, 
  const GrB_Matrix M, 
  const GrB_Matrix A, 
  const GrB_Matrix B,
  const int sz 
)
{
   int dots = end - start;
   // sz = expected non-zeros per dot 
   /*
   int m = (gridDim.x*blockDim.x)*256/sz;
   int dpt = (nvecs+ gridDim.x*blockDim.x -1)/(gridDim.x*blockDim.x);
   m = dpt < m ? dpt : m;
   
   int dots = (nvecs +m -1)/m; 
   */
   const T_A *__restrict__ Ax = (T_A *)A->x  ;
   const T_B *__restrict__ Bx = (T_B *)B->x  ;
   T_C *__restrict__ Cx = (T_C *)C->x  ;
   int64_t *__restrict__ Ci = C->i ;
   const int64_t *__restrict__ Mi = M->i ;
   const int64_t *__restrict__ Ai = A->i ;
   const int64_t *__restrict__ Bi = B->i ;
   const int64_t *__restrict__ Ap = A->p ;
   const int64_t *__restrict__ Bp = B->p ;

   int pfirst, plast;

   GB_PARTITION (pfirst, plast, dots, blockIdx.x, gridDim.x ) ;
   /* 
   if( threadIdx.x ==0 )
   {
      printf("block%d %d dots/thrd, start,end = %ld,%ld pf,pl=%d,%d blockDim=%d\n",
               blockIdx.x, (dots + blockDim.x*gridDim.x -1)/(blockDim.x*gridDim.x), 
               start, end, pfirst, plast, blockDim.x);
   }
   __syncthreads();
   */
   

   int zc = 0 ;
     
   int64_t pair_id;

   //for ( int tid= threadIdx.x +blockDim.x*blockIdx.x;
   //          tid < dots;
   //          tid += blockDim.x * gridDim.x)
   for ( int tid = pfirst+ threadIdx.x ;
             tid < plast;
             tid += blockDim.x )
   {

         pair_id = Bucket[ start + tid ]; 

         int64_t i = Mi [pair_id] ;
         int64_t j = Ci [pair_id]>>4 ; 

         int64_t pA       = Ap[i] ;
         int64_t pA_end   = Ap[i+1] ;
         int64_t pB       = Bp[j] ; 
         int64_t pB_end   = Bp[j+1] ; 

         T_A aki;
         T_B bkj;
         T_Z cij ;

         bool cij_exists = false;

         while (pA < pA_end && pB < pB_end)
         {
            int64_t ia = Ai [pA] ;
            int64_t ib = Bi [pB] ;
            if( ia == ib)
            { 
                // A(k,i) and B(k,j) are the next entries to merge
                #if defined ( GB_PHASE_1_OF_2 )
                cij_exists = true ;
                break ;
                #else
                GB_DOT_MERGE ;
                //GB_DOT_TERMINAL (cij) ;         // break if cij == terminal
                pA++ ;
                pB++ ;
                #endif
            }
            else 
            {
                // A(ia,i) appears before B(ib,j)
                pA += ( ia < ib);
                // B(ib,j) appears before A(ia,i)
                pB += ( ib < ia);
            }
         }
         if (cij_exists){
            GB_PUTC ( Ci[pair_id] = i ) ;
            GB_PUTC ( Cx[pair_id] = (T_C)cij ) ;
         }
         else{
            //printf(" %lld, %lld is zombie %d!\n",i,j,zc);
            zc++; 
            GB_PUTC( Ci[pair_id] = GB_FLIP( i ) ) ;
         }
   }
  
   __syncthreads();

   //printf("thd%d zombie count = %d\n",threadIdx.x,zc);
   zc = block_ReduceSum<int , 32>( this_thread_block(), zc); 
   __syncthreads();

   if( threadIdx.x == 0 && zc > 0) {
      //printf("block%d zombie count = %d\n", blockIdx.x, zc);
      atomicAdd( (unsigned long long int*)&(C->nzombies), (unsigned long long int)zc);
      //C->nzombies += (unsigned long long int)zc;
      //printf("blk:%d Czombie = %lld\n", blockIdx.x,C->nzombies);
   }
   
}
