#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// templates/GB_AxB_cuda_dot3_phase2: fill the global buckets
//------------------------------------------------------------------------------

// TODO describe me

#define GB_CUDA_KERNEL


//#include <cstdint>
#include "GB_cuda_buckets.h"
#include "matrix.h"
#include <hip/hip_cooperative_groups.h>
#include "local_cub/block/block_scan.cuh"

using namespace cooperative_groups;

__global__
void AxB_phase2end
        (
                // input, not modified:
                int64_t *__restrict__ nanobuckets,    // array of size 12-blockDim.x-by-nblocks
                const int64_t *__restrict__ blockbucket,    // global bucket count, of size 12*nblocks
                // output:
                const int64_t *__restrict__ bucketp,        // global bucket cumsum, of size 13
                int64_t *__restrict__ bucket,         // global buckets, of size cnz (== mnz)
                const int64_t *__restrict__ offset,         // global offsets, for each bucket
                // inputs, not modified:
                const GrB_Matrix C,            // output matrix
                const int64_t cnz        // number of entries in C and M
        )
{

    //--------------------------------------------------------------------------
    // get C and M
    //--------------------------------------------------------------------------

    // Ci [p] for an entry C(i,j) contains either GB_FLIP(i) if C(i,j) is a
    // zombie, or (k << 4) + bucket otherwise, where C(:,j) is the kth vector
    // of C (j = Ch [k] if hypersparse or j = k if standard sparse), and
    // where bucket is the bucket assignment for C(i,j).  This phase does not
    // need k, just the bucket for each entry C(i,j).

    int64_t *__restrict__ Ci = C->i ;       // for zombies, or bucket assignment
    int64_t *__restrict__ Mp = C->p ;       // for offset calculations
    int64_t mnvec = C->nvec;

    //--------------------------------------------------------------------------
    // load and shift the nanobuckets for this thread block
    //--------------------------------------------------------------------------

    // The taskbucket for this threadblock is an array of size
    // 12-by-blockDim.x, held by row.  It forms a 2D array within the 3D
    // nanobuckets array.
    int64_t *__restrict__ taskbucket = nanobuckets + blockIdx.x * (NBUCKETS * blockDim.x) ;

    //printf("block%d thd%d blockbucket= %ld\n", blockIdx.x, threadIdx.x,
    //                                           blockbucket[blockIdx.x*gridDim.x+blockIdx.x]);

    // Each thread in this threadblock owns one column of this taskbucket, for
    // its set of 12 nanobuckets.  The nanobuckets are a column of length 12,
    // with stride equal to blockDim.x.
    int64_t *__restrict__ nanobucket = taskbucket + threadIdx.x;

    // Each thread loads its 12 nanobucket values into registers.
#define LOAD_NANOBUCKET(bucket)                     \
        int64_t my_bucket_ ## bucket =                  \
            nanobucket [bucket * blockDim.x]        \
         + blockbucket [bucket * gridDim.x + blockIdx.x]\
         + bucketp [bucket] ; 

    LOAD_NANOBUCKET (0) ;
    LOAD_NANOBUCKET (1) ;
    LOAD_NANOBUCKET (2) ;
    LOAD_NANOBUCKET (3) ;
    LOAD_NANOBUCKET (4) ;
    LOAD_NANOBUCKET (5) ;
    LOAD_NANOBUCKET (6) ;
    LOAD_NANOBUCKET (7) ;
    LOAD_NANOBUCKET (8) ;
    LOAD_NANOBUCKET (9) ;
    LOAD_NANOBUCKET (10) ;
    LOAD_NANOBUCKET (11) ;

    // Now each thread has an index into the global set of 12 buckets,
    // held in bucket, of where to place its own entries.

    //--------------------------------------------------------------------------
    // construct the global buckets
    //--------------------------------------------------------------------------

    // The slice for task blockIdx.x contains entries pfirst:plast-1 of M and
    // C, which is the part of C operated on by this threadblock.
    int64_t pfirst, plast ;

    /*
    for ( int tid_global = threadIdx.x + blockIdx.x * blockDim.x ;
              tid_global < (mnvec+7)/8 ;
              tid_global += blockDim.x * gridDim.x)
    */
    int chunk_max= (cnz + chunksize -1)/chunksize;
    for ( int chunk = blockIdx.x;
          chunk < chunk_max;
          chunk += gridDim.x )
    {

        //GB_PARTITION (pfirst, plast, cnz, tid_global, (mnvec+7)/8 ) ;
        pfirst = chunksize * chunk ;
        plast  = GB_IMIN( chunksize * (chunk+1), cnz ) ;

        int chunk_end;
        if ( cnz > chunksize) chunk_end = GB_IMIN(  chunksize,
                                                    cnz - chunksize*(chunk) );
        else chunk_end = cnz;

        // find the first vector of the slice for task blockIdx.x: the
        // vector that owns the entry Ai [pfirst] and Ax [pfirst].
        //kfirst = GB_search_for_vector_device (pfirst, Mp, 0, mnvec) ;

        // find the last vector of the slice for task blockIdx.x: the
        // vector that owns the entry Ai [plast-1] and Ax [plast-1].
        //klast = GB_search_for_vector_device (plast-1, Mp, kfirst, mnvec) ;


        for ( int p = pfirst + threadIdx.x;
              p < pfirst + chunk_end;
              p += blockDim.x )
        {
            // get the entry C(i,j), and extract its bucket.  Then
            // place the entry C(i,j) in the global bucket it belongs to.

            // TODO: these writes to global are not coalesced.  Instead: each
            // threadblock could buffer its writes to 12 buffers and when the
            // buffers are full they can be written to global.
            int ibucket = Ci[p] & 0xF;
            //printf(" thd: %d p,Ci[p] = %ld,%ld,%d\n", threadIdx.x, p, Ci[p], irow );
            switch (ibucket)
            {
                case  0: bucket [my_bucket_0++ ] = p ; Ci[p] = Ci[p] >>4; break ; //unshift zombies
                case  1: bucket [my_bucket_1++ ] = p ; break ;
                case  2: bucket [my_bucket_2++ ] = p ; break ;
                case  3: bucket [my_bucket_3++ ] = p ; break ;
                case  4: bucket [my_bucket_4++ ] = p ; break ;
                case  5: bucket [my_bucket_5++ ] = p ; break ;
                case  6: bucket [my_bucket_6++ ] = p ; break ;
                case  7: bucket [my_bucket_7++ ] = p ; break ;
                case  8: bucket [my_bucket_8++ ] = p ; break ;
                case  9: bucket [my_bucket_9++ ] = p ; break ;
                case 10: bucket [my_bucket_10++] = p ; break ;
                case 11: bucket [my_bucket_11++] = p ; break ;
                default: break;
            }

        }
        //__syncthreads();
    }
}

