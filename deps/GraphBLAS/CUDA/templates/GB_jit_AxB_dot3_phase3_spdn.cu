#include "hip/hip_runtime.h"
//******************************************************************************
//  Sparse dot products in batch form, sparse - dense case. 
//  Each thread in this kernel is responsible for m vector-pairs(x,y), 
//  m = 256/sz, where sz is in {4, 16, 64, 256}
//  We know each non-zero on the sparse side will hit a dense value.
//  Template on <T_C, T_A, T_B, T_X, T_Y, T_Z >
//  Parameters:

//  int64_t start          <- beginning of bucket  
//  int64_t end            <- end of bucket
//  int64_t *Bucket        <- index of each pair in this bucket
//  matrix<T_C> *C         <- C result matrix 
//  matrix<T_C> *M         <- Mask matrix 
//  matrix<T_A> *A         <- A matrix to multiply, sparse 
//  matrix<T_B> *B         <- B matrix to multiply, dense in sparse format? 
//  int sz                 <- size hint for smaller vector
//******************************************************************************
#include <limits>
#include <cstdint>
#include <stdio.h>
#include "mySemiRing.h"
#include "matrix.h"

template< typename T_C, typename T_A, typename T_B, typename T_X, typename T_Y, typename T_Z>
__global__ void AxB_dot3_phase3_spdn
( 
  int64_t start, 
  int64_t end,
  int64_t *Bucket, 
  GrB_Matrix C, 
  GrB_Matrix M, 
  GrB_Matrix A, 
  GrB_Matrix B,
  int sz 
)
{

   T_A *Ax = (T_A*)A->x;
   T_B *Bx = (T_B*)B->x;
   T_C *Cx = (T_C*)C->x;
   int64_t *Ci = C->i;
   int64_t *Mi = M->i;
   int64_t *Ai = A->i;
   int64_t *Bi = B->i;
   int64_t *Ap = A->p;
   int64_t *Bp = B->p;

   // sz = expected non-zeros per dot 
   int m = 256/sz;
   int nvecs = end - start;
   int dpt = nvecs/32;
   m = dpt < m ? dpt : m;
   //if( threadIdx.x ==0)
   //   printf("thd:%d %d dots/thrd, nvecs = %d blockDim=%d\n",threadIdx.x, sz, nvecs, blockDim.x);
   //__syncthreads();
   int dots = (nvecs +m -1)/m; 
   int zc = 0;
     
   for ( int tid= threadIdx.x +blockDim.x*blockIdx.x;
             tid < dots;
             tid += blockDim.x * gridDim.x) {
      int pair_id, im; 
       //if (threadIdx.x ==0)
       //  printf("thd%u pi=%lld\n",tid, start+threadIdx.x); 
       //  __syncthreads();

      for (pair_id = start+tid, im = 0; 
           im < m && pair_id < end;  
           ++im,     pair_id += dots ){

         int64_t i = Mi[pair_id];
         int64_t j = Ci[pair_id] >> 4;
      //if (threadIdx.x ==0)
      //   printf("thd%u i,j=%lld,%lld\n",tid, i,j); 
      //   __syncthreads();
         
     //  printf("thd%d pi=%d xn=%lld yn=%lld\n",tid, pair_id, 
     //                 A->p[i+1]- A->p[i],
     //                 B->p[j+1]- B->p[j]);

         int64_t pA = Ap[i];
         int64_t pA_end   = Ap[i+1];
         int64_t nnzA   = pA_end - pA;
         int64_t pB = Bp[i];
         int64_t pB_end   = Bp[i+1];
         int64_t nnzB   = pB_end - pB;
         T_A aki;
         T_B bkj;
         T_Z cij;

         if( nnzA == A->vlen) // A is dense
         {
            int64_t k = Bi [pB] ;               // first row index of B(:,j)
            // cij = A(k,i) * B(k,j)
            GB_GETA ( aki=(T_Z)Ax[pA+k] ) ;           // aki = A(k,i)
            GB_GETB ( bkj=(T_Z)Bx[pB] ) ;           // bkj = B(k,j)
            GB_C_MULT ( cij, aki, bkj ) ;           // cij = aki * bkj

            for (int64_t p = pB+1 ; p < pB_end ; p++)
            { 
                //GB_DOT_TERMINAL (cij) ;             // break if cij == terminal
                int64_t k = Bi [p] ;                // next row index of B(:,j)
                // cij += A(k,i) * B(k,j)
                GB_GETA ( aki=(T_Z)Ax[pA+k] ) ;           // aki = A(k,i)
                GB_GETB ( bkj=(T_Z)Bx[p] ) ;           // bkj = B(k,j)
                GB_MULTADD ( cij, aki, bkj ) ;        // cij += aki * bkj
            }

         }
         if( nnzB == B->vlen) // B is dense
         {
            int64_t k = Ai [pA] ;               // first row index of A(:,i)
            // cij = A(k,i) * B(k,j)
            GB_GETA ( aki=(T_Z)Ax[ pA ] ) ;           // aki = A(k,i)
            GB_GETB ( bkj=(T_Z)Bx[ pB+k ] ) ;           // bkj = B(k,j)
            GB_C_MULT ( cij, aki, bkj) ;           // cij = aki * bkj

            for (int64_t p = pA+1 ; p < pA_end ; p++)
            { 
                //GB_DOT_TERMINAL (cij) ;             // break if cij == terminal
                int64_t k = Ai [p] ;                // next row index of A(:,i)
                // cij += A(k,i) * B(k,j)
                GB_GETA ( aki=(T_Z)Ax[ p ] ) ;           // aki = A(k,i)
                GB_GETB ( bkj=(T_Z)Bx[ pB+k] ) ;           // bkj = B(k,j)
                GB_MULTADD ( cij, aki, bkj) ;        // cij += aki * bkj
            }
         }

         GB_PUTC( Ci[pair_id]=i ) ;
         GB_PUTC( Cx[pair_id]=cij ) ;
        
      }
  
   }
   
}
