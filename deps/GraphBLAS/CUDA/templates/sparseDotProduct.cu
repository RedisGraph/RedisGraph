#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// sparseDotProduct_merge_path.cu 
//------------------------------------------------------------------------------

// The sparseDotProduct CUDA kernel produces the semi-ring dot product of two
// sparse vectors of types T1 and T2 and common index space size n, to a scalar 
// odata of type T3. The vectors are sparse, with different numbers of non-zeros.
// ie. we want to produce dot(x,y) in the sense of the given semi-ring.

// This version uses a merge-path algorithm, when the sizes g_xnz and g_ynz are 
// relatively close in size, but for any size of N.

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x. s= 32 with a variable number
// of active threads = min( min(g_xnz, g_ynz), 32) 

// Thus, threadblock b owns a part of the index set spanned by g_xi and g_yi.  Its job
// is to find the intersection of the index sets g_xi and g_yi, perform the semi-ring dot
// product on those items in the intersection, and finally reduce this data to a scalar, 
// on exit write it to g_odata [b].

#include <limits>
#include <hip/hip_cooperative_groups.h>
#include "mySemiRing.h"

using namespace cooperative_groups;

template< typename T, int tile_sz>
__device__ T reduce_sum(thread_block_tile<tile_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        val = ADD( val, g.shfl_down(val,i) );
        //if (g.thread_rank() ==0)
        //    printf("in reduce_sum i=%i val = %f\n", i, val);
    }
    return val; // note: only thread 0 will return full sum
}

#define INTMIN( A, B) ( (A) < (B) ) ?  (A) : (B)
#define INTMAX( A, B) ( (A) > (B) ) ?  (A) : (B)
#define intersects_per_thread 4

template< typename T1, typename T2, typename T3>
__global__ void sparseDotProduct
(
    unsigned int g_xnz,       // Number of non-zeros in x
    unsigned int *g_xi,       // Non-zero indices in x, size xnz
    T1 *g_xdata,              // array of size xnz, type T1
    unsigned int g_ynz,       // Number of non-zeros in y
    unsigned int *g_yi,       // Non-zero indices in y, size ynz
    T2 *g_ydata,              // array of size ynz, type T2
    T3 *g_odata               // array of size grid.x, type T3
)
{
    // set thread ID
    unsigned int tid_global = threadIdx.x+ blockDim.x* blockIdx.x;
    unsigned int tid = threadIdx.x;

    unsigned long int b = blockIdx.x ;

    // total items to be inspected
    unsigned int nxy = (g_xnz + g_ynz);

    //largest possible number of intersections is the smaller nz
    unsigned int n_intersect = INTMIN( g_xnz, g_ynz); 

    //we want more than one intersection per thread
    unsigned int parts = (n_intersect+ intersects_per_thread -1)/ intersects_per_thread; 

    unsigned int work_per_thread = (nxy +parts -1)/parts;
    unsigned int diag = INTMIN( work_per_thread*tid_global, nxy);
    unsigned int diag_end = INTMIN( diag + work_per_thread, nxy);
    //printf(" thd%d parts = %u wpt = %u diag, diag_end  = %u,%u\n",tid, parts, work_per_thread, diag, diag_end); 

   unsigned int x_min = INTMAX( (int)(diag - g_ynz), 0);
   unsigned int x_max = INTMIN( diag, g_xnz);

   //printf("start thd%u x_min = %u x_max = %u\n", tid_global, x_min,x_max);
   while ( x_min < x_max) { //binary search for correct diag break
      unsigned int pivot = (x_min +x_max)/2;
      if ( g_xi[pivot] < g_yi[ diag -pivot -1]) {
         x_min = pivot +1;
      }
      else {
         x_max = pivot;
      }
   }
   int xcoord = x_min;
   int ycoord = diag -x_min -1;
   if (( diag > 0) &&(diag < (g_xnz+g_ynz)) && (g_xi[xcoord] == g_yi[ycoord]) ) { 
       diag--; //adjust for intersection incrementing both pointers 
   }
   // two start points are known now
   int x_start = xcoord;
   int y_start = diag -xcoord; 

   //if (x_start != y_start)
   //   printf("start thd%u  xs,ys = %i,%i\n", tid_global, x_start, y_start);

   x_min = INTMAX( (int)(diag_end - g_ynz), 0);
   x_max = INTMIN( diag_end, g_xnz);

   while ( x_min < x_max) {
      unsigned int pivot = (x_min +x_max)/2;
      //printf("thd%u pre_sw piv=%u diag_e = %u  xmin,xmax=%u,%u\n", tid_global, pivot, diag_end,x_min, x_max);
      if ( g_xi[pivot] < g_yi[ diag_end -pivot -1]) {
         x_min = pivot +1;
      }
      else {
         x_max = pivot;
      }
      //printf("thd%u piv=%u xmin,xmax = %u,%u\n", tid_global, pivot, x_min, x_max);
   }
   xcoord = x_min;
   ycoord = diag_end -x_min -1;
   if ( (diag_end < (g_xnz+g_ynz)) && (g_xi[xcoord] == g_yi[ycoord]) ) { 
       diag--; //adjust for intersection incrementing both pointers  
   }
   // two end points are known now
   int x_end = xcoord; 
   int y_end = diag_end - xcoord; 

   /* 
   if (tid == 0 && b == 0) {
        printf ("type1 is size %d\n", sizeof (T1)) ;
        for (int k = 0 ; k < g_xnz ; k++) printf ("%4d: %g,", k, (T1) g_xdata [k]) ;
        printf ("\n") ;
        printf ("type2 is size %d\n", sizeof (T2)) ;
        for (int k = 0 ; k < g_ynz ; k++) printf ("%4d: %g,", k, (T2) g_ydata [k]) ;
        printf ("\n") ;
    }
    __syncthreads();
    */

    T3 sum = (T3) 0;
    //printf(" thd%u has init value %f\n",tid, sum);

    // nothing to do
    if ( (x_start >= x_end) || (y_start >= y_end) ) { return ; }

    //merge-path dot product
    int k = x_start;
    int l = y_start;
    while ( k < x_end && l < y_end )
    {
       if      ( g_xi[k] < g_yi[l] ) k += 1;
       else if ( g_xi[k] > g_yi[l] ) l += 1; 
       else {
          //printf("  thd%d ix at %u \n",tid_global,g_xi[k]);
          //printf("   sum += %f * %f \n",tid,g_xdata[k],g_ydata[l]);
          //sum = ADD( sum, MUL( g_xdata[k], g_ydata[l]));
          MULADD( sum, g_xdata[k], g_ydata[l]);
          //printf(" thd%u work value = %f\n",tid_global, sum);
          k+= 1;
          l+= 1;
       }

    }

    __syncthreads ( ) ;
    /*
    if (1)
    {
        printf ("thd%u done with intersect and multiply, val = %f\n",tid_global, sum) ;
    }
    __syncthreads ( ) ;
    */

    //--------------------------------------------------------------------------
    // reduce sum per-thread values to a single scalar
    //--------------------------------------------------------------------------
    // Using tile size fixed at compile time, we don't need shared memory
    #define tile_sz 32 
    thread_block_tile<tile_sz> tile = tiled_partition<tile_sz>( this_thread_block());
    T3 block_sum = reduce_sum<T3,tile_sz>(tile, sum);

    // write result for this block to global mem
    if (tid == 0)
    {
        printf ("final %d : %g\n", b,  block_sum) ;
        g_odata [b] = block_sum ;
    }
}

