#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// reduceUnrolled.cu
//------------------------------------------------------------------------------

// The reduceUnrolled CUDA kernel reduces an array g_idata of size n, of any
// type T, to an array g_odata of size grid.x.  Each threadblock (blockIdx.x)
// reduces its portion of g_idata to a single scalar, g_odata [blockIdx.x].

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x.
// Each threadblock owns s*8 contiguous items in the input data.

// Thus, threadblock b owns g_idata [b*s*8 ... min(n,(b+1)*s*8-1)].  It's job
// is to reduce this data to a scalar, and write it to g_odata [b].

#include "myOp.h"
#include <hip/hip_cooperative_groups.h>
#include "GB_cuda.h"

GrB_Matrix Stuff ;  // hack hack hack

using namespace cooperative_groups;

template< typename T, int tile_sz>
__inline__ __device__ 
T warp_ReduceSum( thread_block_tile<tile_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        T fold = g.shfl_down( val, i);
        //printf("thd%d   %d OP %d is %d\n", threadIdx.x, val, fold, OP( val, fold));
        val = OP( val, fold );
    }
    //if (threadIdx.x ==0) printf("thd%d single warp sum is %d\n", threadIdx.x,  val);
    return val; // note: only thread 0 will return full sum
}

template<typename T, int warpSize>
__inline__ __device__
T block_ReduceSum(thread_block g, T val)
{
  static __shared__ T shared[warpSize]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;
  thread_block_tile<warpSize> tile = tiled_partition<warpSize>( g );

  // Each warp performs partial reduction
  val = warp_ReduceSum<T, warpSize>( tile, val);    

  // Wait for all partial reductions
  if (lane==0) { 
     //printf("thd%d warp%d sum is %d\n", threadIdx.x, wid, val);
     shared[wid]=val; // Write reduced value to shared memory
     //printf("thd%d stored warp %d sum %d\n", threadIdx.x, wid, val);
  }
  g.sync();              // Wait for all partial reductions

  if (wid > 0 || gridDim.x == 1 ) return val;
  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : MONOID_IDENTITY;
  //printf("thd%d warp loaded val = %d\n", threadIdx.x, lane, val);

  
  if (wid==0) val = warp_ReduceSum<T, warpSize>( tile, val); //Final reduce within first warp

  return val;
}

template< typename T>
__global__ void reduceUnrolled
(
    T *g_idata,     // array of size n
    T *g_odata,     // array of size grid.x
    unsigned int n
)
{
    // set thread ID
    unsigned int tid = threadIdx.x ;

    // this threadblock b owns g_idata [block_start ... block_end-1]
    unsigned long int s = blockDim.x ;
    unsigned long int b = blockIdx.x ;
    unsigned long int block_start = b * s * 8 ;
    unsigned long int block_end   = (b + 1) * s * 8 ;

    /*
    if (tid == 0)
    {
        printf ("block %d: [%lu ... %ld]\n", b, block_start, block_end-1) ;
    }
    */

    /*
    if (tid == 0 && b == 0)
    {
        printf ("type is size %d\n", sizeof (T)) ;
        for (int k = 0 ; k < n ; k++) printf ("%4d: %g\n", k, (double) g_idata [k]) ;
        printf ("\n") ;
    }
    */

    // nothing to do
    if (block_start > block_end) { if (tid == 0) printf ("bye!\n") ; return ; }

    // convert global data pointer to the local pointer of this block
    T *idata = g_idata + block_start ;

    T x0, x1, x2, x3, x4, x5, x6, x7 ;

    if (block_end <= n)
    {
        // unrolling 8
        x0 = idata [tid] ;
        x1 = idata [tid +     s] ;
        x2 = idata [tid + 2 * s] ;
        x3 = idata [tid + 3 * s] ;
        x4 = idata [tid + 4 * s] ;
        x5 = idata [tid + 5 * s] ;
        x6 = idata [tid + 6 * s] ;
        x7 = idata [tid + 7 * s] ;

        /*
        if (b == 0)
        {
            printf ("block zero: here is tid %2d : %g %g %g %g %g %g %g %g \n", tid,
                (double) x0, (double) x1, (double) x2, (double) x3,
                (double) x4, (double) x5, (double) x6, (double) x7) ;
        }
        */

    }
    else
    {
        // the last block has size less than 8*s
        #define IDATA(i) ((i < lastblocksize) ? idata [i] : MONOID_IDENTITY)
        int lastblocksize = n - block_start ;
        x0 = IDATA (tid) ;
        x1 = IDATA (tid +     s) ;
        x2 = IDATA (tid + 2 * s) ;
        x3 = IDATA (tid + 3 * s) ;
        x4 = IDATA (tid + 4 * s) ;
        x5 = IDATA (tid + 5 * s) ;
        x6 = IDATA (tid + 6 * s) ;
        x7 = IDATA (tid + 7 * s) ;
    }
    T sum;
    //work [tid] = x0 + x1 + x2 + x3 + x4 + x5 + x6 + x7 ;
    sum = OP( x0 ,OP( x1, OP( x2, OP( x3,
                 OP( x4, OP( x5 , OP( x6 , x7))))))) ;

        /*
        if (b == 0)
        {
            printf ("block zero: still is tid %2d : %g %g %g %g %g %g %g %g \n", tid,
                (double) x0, (double) x1, (double) x2, (double) x3,
                (double) x4, (double) x5, (double) x6, (double) x7) ;
        }

        if (b == 0)
        {
            printf ("block zero: here is tid %d result %g  is %g\n",
            tid, (double) work [tid],
            (double) (x0 + x1 + x2 + x3 + x4 + x5 + x6 + x7)) ;
        }
        */

    __syncthreads ( ) ;

    //--------------------------------------------------------------------------
    // reduce work [0..s-1] to a single scalar
    //--------------------------------------------------------------------------

    // This assumes that s is a power of 2 and <= 1024, and at least 32
    // This assumes blockDim is a multiple of 32
    sum = block_ReduceSum<T , 32>( this_thread_block(), sum); 

    // write result for this block to global mem
    if (tid == 0)
    {
        // printf ("final %d : %g\n", b, (double) work [0]) ;
        g_odata [b] = sum ;
    }
}

