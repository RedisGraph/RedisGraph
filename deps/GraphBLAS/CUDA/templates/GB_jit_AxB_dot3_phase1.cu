#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// templates/GB_AxB_cuda_dot3_phase1: symbolic load balancing and data partition
// to assign work to different 'buckets' for later compute
//------------------------------------------------------------------------------

//  This kernel scans the non-zero pattern in A and B, takes into account the
//  mask and computes total work required to form C. Then it classifies each
//  dot product into a set of buckets for efficient compute. 

#define GB_KERNEL
#include <limits>
#include <cstdint>
#include "matrix.h"
#include "GB_cuda_buckets.h"
#include "local_cub/block/block_scan.cuh"
#include "mySemiRing.h"

//------------------------------------------------------------------------------
// GB_bucket_assignment
//------------------------------------------------------------------------------

// assign the dot product C(i,j) = A(:,i)'*B(:,j) to a specific bucket
__device__ static inline GB_bucket_code GB_bucket_assignment
(
    int64_t ainz,       // # of entries A(:,i), always > 0
    int64_t bjnz,       // # of entries B(:,j), always > 0
    int64_t vlen        // vector length of A(:,i) and B(:,j)
)
{

    int b = 0 ; // no bucket assigned yet

    // GB_BUCKET (condition,bucket) :  assigns an entry to a bucket,
    // if the condition holds, but without using any if statements.
    // An entry is assigned once and not reassigned.

    // If the bucket b has not assigned, it is b = 0.  The GB_BUCKET function
    // tests this case, and if the condition is also true, the expression
    // (b==0) * condition * (bucket+1) becomes equal to bucket+1.  This
    // value is added to b, which is zero, so the final result is that b
    // is set to bucket+1.

    // If the bucket b has been assigned already, we have b > 0.  Thus,
    // the expression ((b==0) * condition * (bucket+1)) becomes zero.
    // When added to b, the result is that b doesn't change, so the bucket
    // assignment b is unmodified.

    #define GB_BUCKET(condition,bucket) \
        b = (((b == 0) * (condition)) * (bucket+1)) + b ;

//  if (ia_last < ib_first || ib_last < ia_first)
    { 

        //----------------------------------------------------------------------
        // pattern of A(:,i) and B(:,j) do not overlap
        //----------------------------------------------------------------------

        // The patterns of A(:,i) and B(:,j) are always sorted.  If the last
        // entry in A(:,i) comes before the first entry in B(:,j), or visa
        // versa, then there is no work to do since C(i,j) must be a zombie.

        // GB_BUCKET (ia_last < ib_first || ib_last < ia_first, GB_BUCKET_ZOMBIE);

    }
//  else if (bjnz == vlen && ainz == vlen && vlen > 256)
    {

        //----------------------------------------------------------------------
        // both A(:,i) and B(:,j) are dense
        //----------------------------------------------------------------------

        // No search of A(:,i) or B(:,j) is needed.  Total work is O(vlen).
        // The intersection is non-empty, so C(i,j) cannot be a zombie.

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_dndn.cu.jit

        GB_BUCKET (bjnz == vlen && ainz == vlen && vlen > 256, GB_BUCKET_DNDN) ;

    }
//  else if (ainz == vlen)
    {
 
        //----------------------------------------------------------------------
        // A(:,i) is dense and B(:,j) is sparse
        //----------------------------------------------------------------------
 
        // No search of A(:,i) is needed.  Total work is O(bjnz), via a linear
        // time scan of B(:,j).  Since A(:,i) is dense and B(:,j) is non-empty,
        // the intersection is non-empty, so C(i,j) cannot be a zombie.

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_spdn.cu.jit
        // Two buckets are used, depending on bjnz.
        GB_BUCKET (ainz == vlen && bjnz <  256, GB_BUCKET_DNVS) ;
        GB_BUCKET (ainz == vlen && bjnz >= 256, GB_BUCKET_DNSP) ;
 
    }
//  else if (bjnz == vlen)
    {

        //----------------------------------------------------------------------
        // A(:,i) is sparse and B(:,j) is dense
        //----------------------------------------------------------------------

        // No search of B(:,j) is needed.  Total work is O(ainz), via a linear
        // time scan of A(:,i).  Since B(:,j) is dense and A(:,i) is non-empty,
        // the intersection is non-empty, so C(i,j) cannot be a zombie.

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_spdn.cu.jit
        // Two buckets are used, depending on ainz.
        GB_BUCKET (bjnz == vlen && ainz <  256, GB_BUCKET_VSDN) ;
        GB_BUCKET (bjnz == vlen && ainz >= 256, GB_BUCKET_SPDN) ;

    }
//  else if ((ainz > 32 * bjnz && bjnz < 256)
//        || (bjnz > 32 * ainz && ainz < 256))
    {

        //----------------------------------------------------------------------
        // A(:,i) is very sparse compared to B(:,j), or visa versa
        //----------------------------------------------------------------------

        // Since B(:,j) is small, and much smaller than A(:,i), the efficient
        // way to compute C(i,j) is a linear scan of B(:,j).  For each B(k,j),
        // a binary search for the index A(k,i) is done.  The expected work to
        // compute C(i,j) is thus O(bjnz * log2 (ainz)).  If A(:,i) is very
        // sparse compared to B(:,j), the opposite is done inside the kernel.

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_vssp.cu.jit

        GB_BUCKET ((ainz > 32 * bjnz && bjnz < 256)
                || (bjnz > 32 * ainz && ainz < 256), GB_BUCKET_VSSP) ;

    }
//  else if (ainz + bjnz <= 4)
    {

        //----------------------------------------------------------------------
        // both A(:,i) and B(:,j) are very tiny (total size 4 or less)
        //----------------------------------------------------------------------

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_vsvs.cu.jit
        //GB_BUCKET (ainz + bjnz <= 4, GB_BUCKET_VSVS_4) ;

    }
//  else if (ainz + bjnz <= 16)
    {

        //----------------------------------------------------------------------
        // both A(:,i) and B(:,j) are tiny (total size 16 or less)
        //----------------------------------------------------------------------

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_vsvs.cu.jit
        //GB_BUCKET (ainz + bjnz <= 16, GB_BUCKET_VSVS_16) ;

    }
//  else if (ainz + bjnz <= 64)
    {

        //----------------------------------------------------------------------
        // both A(:,i) and B(:,j) are small (total size 64 or less)
        //----------------------------------------------------------------------

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_vsvs.cu.jit
        //GB_BUCKET (ainz + bjnz <= 64, GB_BUCKET_VSVS_64) ;

    }
//  else if (ainz + bjnz <= 256)
    {

        //----------------------------------------------------------------------
        // both A(:,i) and B(:,j) are modest in size (total size 256 or less)
        //----------------------------------------------------------------------

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_vsvs.cu.jit
        GB_BUCKET (ainz + bjnz <= 256, GB_BUCKET_VSVS_256) ;

    }
//  else
    {

        //----------------------------------------------------------------------
        // default: use the merge-path method
        //----------------------------------------------------------------------

        // CUDA kernel: templates/GB_jit_AxB_dot3_phase3_mp.cu.jit
        GB_BUCKET (true, GB_BUCKET_MERGEPATH) ;
    }

    // subtract one to undo the "bucket+1" assignment in the
    // GB_BUCKET macro assignment expression.
    return (GB_bucket_code) (b-1) ;
}


//--------------------------------------------------------------------------
// GB_AxB_cuda_dot3_phase1: build nanobuckets, hunt for pre-zombies
//--------------------------------------------------------------------------

// GB_AxB_cuda_dot3_phase1 is a CUDA kernel that scans all entries in C and
// assigns them to each of the 12 buckets.  The output is a 12-by-blockDim array of
// bucket counts, per threadblock (the nanobucket array).  Each of the blockDim.x 
// threads has its own set of 12 bucket counts.  Each threadblock in this
// kernel then computes the first part of the cumulative sum of the
// nanobuckets, and writes it to global memory.

// The kernel also computes Ci, of size nnz(C), which contains the
// zombie assignment or bucket assignment for non-zombies in C.

template<typename Type_M> 
__global__ void GB_AxB_cuda_dot3_phase1
(
    // outputs, preallocated in global memory:
    int64_t *nanobuckets,       // array of size 12-blockDim.x-by-gridDim.x
    int64_t *blockbucket,       // bucket counts, of size 12-by-gridDim.x
    // input/output:
    GrB_Matrix C,               // final output matrix
    // inputs, not modified:
    const GrB_Matrix M,         // mask matrix
    const GrB_Matrix A,         // input matrix
    const GrB_Matrix B          // input matrix
)
{

    //--------------------------------------------------------------------------
    // get C, M, A, and B
    //--------------------------------------------------------------------------
    
    const int64_t *__restrict__ Mh = M->h ;
    const int64_t *__restrict__ Mp = M->p ;
    const int64_t *__restrict__ Mi = M->i ;
    const Type_M *__restrict__ Mx = (Type_M*)M->x ;    // not accessed if M is structural
    const int64_t mnvec = M->nvec ;
    const int64_t mnz =  GB_NNZ(M) ;
    const bool M_is_hyper = M->is_hyper ;

    const int64_t *__restrict__ Ah = A->h ;
    const int64_t *__restrict__ Ap = A->p ;
    const int64_t *__restrict__ Ai = A->i ;
    const int64_t avlen = A->vlen ;
    const int64_t anz = GB_NNZ(A) ;
    const bool A_is_hyper = A->is_hyper ;

    const int64_t *__restrict__ Bh = B->h ;
    const int64_t *__restrict__ Bp = B->p ;
    const int64_t *__restrict__ Bi = B->i ;
    const int64_t bvlen = B->vlen ;
    const int64_t bnz = GB_NNZ(B);
    const bool B_is_hyper = B->is_hyper ;

    // int64_t *restrict Cp = C->p ;    // copy of Mp
    // int64_t *restrict Ch = C->h ;    // copy of Mh
    int64_t *__restrict__ Ci = C->i ;       // for zombies, or bucket assignment

    // Ci [p] for an entry C(i,j) contains either GB_FLIP(i) if C(i,j) is a
    // zombie, or (k << 4) + bucket otherwise, where C(:,j) is the kth vector
    // of C (j = Ch [k] if hypersparse or j = k if standard sparse), and
    // where bucket is the bucket assignment for C(i,j). 
    // bucket can be recovered from Ci by bucket = Ci & 0xF

    //--------------------------------------------------------------------------
    // clear the bucket counters
    //--------------------------------------------------------------------------

    //ASSERT (mnz > 0) ;
    //ASSERT (gridDim.x <= mnz) ;

    // each thread uses 12 bucket counters, held in register
    int64_t my_bucket_0  = 0 ;
    int64_t my_bucket_1  = 0 ;
    int64_t my_bucket_2  = 0 ;
    int64_t my_bucket_3  = 0 ;
    int64_t my_bucket_4  = 0 ;
    int64_t my_bucket_5  = 0 ;
    int64_t my_bucket_6  = 0 ;
    int64_t my_bucket_7  = 0 ;
    int64_t my_bucket_8  = 0 ;
    int64_t my_bucket_9  = 0 ;
    int64_t my_bucket_10 = 0 ;
    int64_t my_bucket_11 = 0 ;

    // Registers cannot be indexed (!) so this macro is used instead.
    // The bucket registers are indexed by the GB_bucket_code enum.
    #define GB_BUCKET_COUNT(bucket)                 \
    {                                               \
        switch (bucket)                             \
        {                                           \
            case  0: my_bucket_0++  ; break ;       \
            case  1: my_bucket_1++  ; break ;       \
            case  2: my_bucket_2++  ; break ;       \
            case  3: my_bucket_3++  ; break ;       \
            case  4: my_bucket_4++  ; break ;       \
            case  5: my_bucket_5++  ; break ;       \
            case  6: my_bucket_6++  ; break ;       \
            case  7: my_bucket_7++  ; break ;       \
            case  8: my_bucket_8++  ; break ;       \
            case  9: my_bucket_9++  ; break ;       \
            case 10: my_bucket_10++ ; break ;       \
            case 11: my_bucket_11++ ; break ;       \
        }                                           \
    }
     /*
    if(threadIdx.x==0 ) {
       printf(" in phase1 kernel, mnz,anz,bnz= %ld,%ld,%ld\n",mnz,anz,bnz); 
    }
    __syncthreads();
     */
     #define pointerchunk 256

     __shared__ int64_t Mps[pointerchunk];
     __shared__ int64_t ks [chunksize];

    //--------------------------------------------------------------------------
    // compute the task descriptor
    //--------------------------------------------------------------------------

    // all threads in this block will compute the same values for these:
    int32_t pfirst, plast, kfirst, klast ;
    /*
    for ( int tid_global = threadIdx.x + blockIdx.x * blockDim.x ; 
              tid_global < (mnvec+ 7)/8 ;
              tid_global += blockDim.x*gridDim.x) 
              */
    int chunk_max= (mnz + chunksize -1)/chunksize;
    for ( int chunk = blockIdx.x;
              chunk < chunk_max;
              chunk += gridDim.x ) 
    {

      // The slice for each task contains entries pfirst:plast-1 of M and C.
      //GB_PARTITION (pfirst, plast, mnz, chunk, (mnz+1023)/1024 ) ;
      pfirst = chunksize * chunk ; 
      plast  = GB_IMIN( chunksize * (chunk+1), mnz ) ;

      int chunk_end;
      if ( mnz > chunksize) chunk_end = GB_IMIN(  chunksize, 
                                                  mnz - chunksize*(chunk) ) ; 
      else chunk_end = mnz;

      // find the first vector of the slice for task tid_global: the
      // vector that owns the entry Ai [pfirst] and Ax [pfirst].
      kfirst = GB_search_for_vector_device (pfirst, Mp, 0, mnvec) -1 ;
      //if( pfirst ==0) kfirst = 0;

      // find the last vector of the slice for task blockIdx.x: the
      // vector that owns the entry Ai [plast-1] and Ax [plast-1].
      klast = GB_search_for_vector_device (plast-1, Mp, kfirst, mnvec) ;

      int k_end = GB_IMIN(  pointerchunk ,  klast - kfirst +2 ) ;
       /* 
      if( threadIdx.x ==0) 
      {
         printf("chunk%d pfirst,plast,ch_end =%d,%d,%d kfirst,klast,kend = %d,%d,%d\n",
                 chunk, pfirst, plast, chunk_end, kfirst, klast, k_end ) ;
      }
      __syncthreads();
      */
      
     
      // load pointer values for this chunk
      for ( int i = threadIdx.x; i< k_end; i+= blockDim.x)
      {
          Mps[i] = Mp[i + kfirst];
      }
      __syncthreads();

      // search for k values for each entry
      float slope = (float)(mnvec)/(float)(mnz* chunksize) ;
      for ( int i =  threadIdx.x; i< chunk_end; i+= blockDim.x)
      {   
          ks[i] = kfirst + slope*( float )(i);
          while ( Mps[ ks[i] - kfirst + 1 ] <= (i+pfirst) )
             ks[i]++;
          while ( Mps[ ks[i] - kfirst     ] >  (i+pfirst) )
             ks[i]--;
      }
      __syncthreads();


    //ASSERT (0 <= kfirst && kfirst <= klast && klast < mnvec) ;
    /*
    if (threadIdx.x ==0 ) {
       printf ("threadblock %d  after ksearch pfirst %ld plast %ld kfirst %ld klast %ld\n",
                blockIdx.x, pfirst, plast, kfirst, klast) ;
    }
    __syncthreads();
    */
    
    

    //--------------------------------------------------------------------------
    // assign entries in C(i,j) to the buckets
    //--------------------------------------------------------------------------

    // if B is hypersparse, bpleft ... TODO describe
    // int64_t bpleft = 0 ;
    
        //----------------------------------------------------------------------
        // no binary search variant
        //----------------------------------------------------------------------

        //printf ("no binary search\n") ;

        //int32_t pM_start, pM_end ;
        //for (int64_t pM = pfirst + threadIdx.x ; pM < plast ; pM += blockDim.x)
        int32_t i,j;
        int32_t k = kfirst ;
            
        //for (int64_t pM = pfirst; pM < plast; pM++ ) 
        for ( int pM = pfirst + threadIdx.x;
                  pM < pfirst + chunk_end;
                  pM += blockDim.x )
        {
            GB_bucket_code bucket = GB_BUCKET_ZOMBIE ;
            k = ks[ pM - pfirst ] ;
            //k += ( pM == Mp[k+1] ) ;
            //printf ("tid%d  k %ld pM %ld\n", tid_global, k, pM;
            i = Mi [ pM ] ;

            if ( MX ( pM ) )
            { 

            // do a binary search for k (and j) that has this entry M(i,j)
            //k = GB_search_for_vector_device (pM, Mp, k, klast) ;

// HACK
j = k ;
//          int64_t j = (Mh == NULL) ? k : Mh [k] ;

            //--------------------------------------------------------------
            // get B(:,j)
            //--------------------------------------------------------------

            int64_t pB, pB_end ;
// HACK: for sparse only, not hypersparse
pB = Bp [j] ;
pB_end = Bp [j+1] ;
//              GB_lookup_device (B_is_hyper, Bh, Bp, &bpleft, bnvec-1, j,
//                  &pB, &pB_end) ;
                int64_t bjnz = pB_end - pB ;
                if (bjnz > 0)
                {
                 //   int64_t ib_first = Bi [pB] ;
                 //   int64_t ib_last  = Bi [pB_end-1] ;

                    //----------------------------------------------------------
                    // get A(:,i)
                    //----------------------------------------------------------

                    int64_t pA, pA_end ;
                    //int64_t apleft = 0 ;
// HACK: for sparse only, not hypersparse
pA = Ap [i] ;
pA_end = Ap [i+1] ;
//                  GB_lookup_device (A_is_hyper, Ah, Ap, &apleft, anvec-1, i,
//                      &pA, &pA_end) ;
                    int64_t ainz = pA_end - pA ;
                    if (ainz > 0)
                    {
                     //   int64_t ia_first = Ai [pA] ;
                     //   int64_t ia_last  = Ai [pA_end-1] ;

                        //------------------------------------------------------
                        // determine the bucket for C(i,j)
                        //------------------------------------------------------

                        //bucket = GB_BUCKET_MERGEPATH ;
                         bucket= GB_bucket_assignment ( ainz, bjnz, bvlen) ;
                    }
                }
            }

            if (bucket == GB_BUCKET_ZOMBIE)
            {
                // mark C(i,j) is a zombie
                //printf ("tid%d pM=%d %d,%d prezombie\n",threadIdx.x,pM,i,j) ;
                Ci [pM] = GB_FLIP (i) << 4 ;
                // GB_BUCKET_COUNT (GB_BUCKET_ZOMBIE) ;
                my_bucket_0++ ; //0 is the zombie bucket
            }
            else
            {
                // place C(i,j) in its bucket
                Ci [pM] = (k << 4) + bucket ;
                GB_BUCKET_COUNT (bucket) ;
                //printf ("tid%d pM=%d %d,%d b=%d\n",threadIdx.x, pM, i,j, (int)bucket) ;
            }
         }
            
        
    
    }
    __syncthreads();

    //--------------------------------------------------------------------------
    // cumulative sum of each bucket
    //--------------------------------------------------------------------------

    typedef hipcub::BlockScan<int64_t, 32, hipcub::BLOCK_SCAN_WARP_SCANS> BlockCumSum; 
    __shared__ typename BlockCumSum::TempStorage temp_storage;

    // The taskbucket for this thread block is an array of size
    // 12-by-blockDim.x, held by row.  Each thread owns one column of this
    // taskbucket, the nanobucket.  The nanobucket is a column of length 12,
    // with stride equal to blockDim.x.
    int64_t *nanobucket =
        nanobuckets + blockIdx.x * (12 * blockDim.x) + threadIdx.x ;

    #define CUMSUM_AND_STORE_NANOBUCKET(bucket)                             \
        if( threadIdx.x == blockDim.x-1)                                    \
            blockbucket [blockIdx.x + bucket * gridDim.x] =                 \
            my_bucket_ ## bucket ;                                          \
        BlockCumSum(temp_storage).ExclusiveSum                              \
            ( my_bucket_ ## bucket, my_bucket_ ## bucket) ;                 \
            __syncthreads();                                                \
        nanobucket [bucket * blockDim.x] = my_bucket_ ## bucket ;

    CUMSUM_AND_STORE_NANOBUCKET (0) ;
    CUMSUM_AND_STORE_NANOBUCKET (1) ;
    CUMSUM_AND_STORE_NANOBUCKET (2) ;
    CUMSUM_AND_STORE_NANOBUCKET (3) ;
    CUMSUM_AND_STORE_NANOBUCKET (4) ;
    CUMSUM_AND_STORE_NANOBUCKET (5) ;
    CUMSUM_AND_STORE_NANOBUCKET (6) ;
    CUMSUM_AND_STORE_NANOBUCKET (7) ;
    CUMSUM_AND_STORE_NANOBUCKET (8) ;
    CUMSUM_AND_STORE_NANOBUCKET (9) ;
    CUMSUM_AND_STORE_NANOBUCKET (10) ;
    CUMSUM_AND_STORE_NANOBUCKET (11) ;

    /*    
    if(threadIdx.x +blockIdx.x*blockDim.x <= mnvec) //blockDim.x -1){ 
    {
       printf("thd %d blk%d nbucket0 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[0]);
       printf("thd %d blk%d nbucket1 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[1*blockDim.x]);
       printf("thd %d blk%d nbucket2 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[2*blockDim.x]);
       printf("thd %d blk%d nbucket3 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[3*blockDim.x]);
       printf("thd %d blk%d nbucket4 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[4*blockDim.x]);
       printf("thd %d blk%d nbucket5 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[5*blockDim.x]);
       printf("thd %d blk%d nbucket6 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[6*blockDim.x]);
       printf("thd %d blk%d nbucket7 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[7*blockDim.x]);
       printf("thd %d blk%d nbucket8 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[8*blockDim.x]);
       printf("thd %d blk%d nbucket9 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[9*blockDim.x]);
       printf("thd %d blk%d nbucket10 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[10*blockDim.x]);
       printf("thd %d blk%d nbucket11 has %ld prev\n",threadIdx.x, blockIdx.x, nanobucket[11*blockDim.x]);

    }
    __syncthreads();
    */
        

    // The last thread now has the sum of all nanobuckets, which is then saved
    // to the global bucket counts.   blockbucket is an array of size
    // 12-by-gridDim.x, held by row, with one column per thread block.
    // The last thread saves its result in the column of this thread block.
    // Note that this write to global memory is not coalesced.

    #define STORE_GLOBAL_BUCKET_COUNT(bucket)                    \
        blockbucket [blockIdx.x + bucket * gridDim.x] +=         \
            my_bucket_ ## bucket ;

    if (threadIdx.x == blockDim.x - 1 ) 
    {
        STORE_GLOBAL_BUCKET_COUNT (0) ;
        STORE_GLOBAL_BUCKET_COUNT (1) ;
        STORE_GLOBAL_BUCKET_COUNT (2) ;
        STORE_GLOBAL_BUCKET_COUNT (3) ;
        STORE_GLOBAL_BUCKET_COUNT (4) ;
        STORE_GLOBAL_BUCKET_COUNT (5) ;
        STORE_GLOBAL_BUCKET_COUNT (6) ;
        STORE_GLOBAL_BUCKET_COUNT (7) ;
        STORE_GLOBAL_BUCKET_COUNT (8) ;
        STORE_GLOBAL_BUCKET_COUNT (9) ;
        STORE_GLOBAL_BUCKET_COUNT (10) ;
        STORE_GLOBAL_BUCKET_COUNT (11) ;
    }
    
    /* 
    if(threadIdx.x == blockDim.x -1){ 

       printf("block%d bbucket0 has %ld entries\n",blockIdx.x, blockbucket[0*gridDim.x+blockIdx.x]);
       printf("block%d bbucket1 has %ld entries\n",blockIdx.x, blockbucket[1*gridDim.x+blockIdx.x]);
       printf("block%d bbucket2 has %ld entries\n",blockIdx.x, blockbucket[2*gridDim.x+blockIdx.x]);
       printf("block%d bbucket3 has %ld entries\n",blockIdx.x, blockbucket[3*gridDim.x+blockIdx.x]);
       printf("block%d bbucket4 has %ld entries\n",blockIdx.x, blockbucket[4*gridDim.x+blockIdx.x]);
       printf("block%d bbucket5 has %ld entries\n",blockIdx.x, blockbucket[5*gridDim.x+blockIdx.x]);
       printf("block%d bbucket6 has %ld entries\n",blockIdx.x, blockbucket[6*gridDim.x+blockIdx.x]);
       printf("block%d bbucket7 has %ld entries\n",blockIdx.x, blockbucket[7*gridDim.x+blockIdx.x]);
       printf("block%d bbucket8 has %ld entries\n",blockIdx.x, blockbucket[8*gridDim.x+blockIdx.x]);
       printf("block%d bbucket9 has %ld entries\n",blockIdx.x, blockbucket[9*gridDim.x+blockIdx.x]);
       printf("block%d bbucket10 has %ld entries\n",blockIdx.x, blockbucket[10*gridDim.x+blockIdx.x]);
       printf("block%d bbucket11 has %ld entries\n",blockIdx.x, blockbucket[11*gridDim.x+blockIdx.x]);

    }
    __syncthreads();
    */
    
}

