#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// AxB_dot3_phase3_mp.cu 
//------------------------------------------------------------------------------

// This CUDA kernel produces the semi-ring product of two
// sparse matrices of types T_A and T_B and common index space size n, to a  
// output matrix of type T_C. The matrices are sparse, with different numbers
// of non-zeros and different sparsity patterns. 
// ie. we want to produce C = A'*B in the sense of the given semi-ring.

// This version uses a merge-path algorithm, when the sizes nnzA and nnzB are 
// relatively close in size, neither is very spare nor dense, for any size of N.
// Handles arbitrary sparsity patterns with guaranteed load balance.

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x. s= 32 with a variable number
// of active threads = min( min(g_xnz, g_ynz), 32) 

// Thus, threadblock b owns a part of the index set spanned by g_xi and g_yi.  Its job
// is to find the intersection of the index sets g_xi and g_yi, perform the semi-ring dot
// product on those items in the intersection, and finally reduce this data to a scalar, 
// on exit write it to g_odata [b].

//  int64_t start          <- start of vector pairs for this kernel
//  int64_t end            <- end of vector pairs for this kernel
//  int64_t *Bucket        <- array of pair indices for all kernels 
//  matrix<T_C> *C         <- result matrix 
//  matrix<T_M> *M         <- mask matrix
//  matrix<T_A> *A         <- input matrix A
//  matrix<T_B> *B         <- input matrix B
#include <limits>
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include "mySemiRing.h"
#include "matrix.h"

// Using tile size fixed at compile time, we don't need shared memory
#define tile_sz 32 

using namespace cooperative_groups;

template< typename T, int warp_sz>
__device__ __inline__ 
T GB_reduce_sum(thread_block_tile<warp_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        T next = g.shfl_down( val, i);
        val = GB_ADD( val, next ) ;
    }
    return val;
}

template< typename T, int warp_sz>
__device__ __inline__ 
T reduce_plus(thread_block_tile<warp_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        val += g.shfl_down( val, i) ;
    }
    return val; // note: only thread 0 will return full sum and flag value
}

#define intersects_per_thread 8

template< typename T_C, typename T_A, typename T_B, typename T_X, typename T_Y, typename T_Z>  
__global__ void AxB_dot3_phase3_mp
(
    int64_t start,
    int64_t end,
    int64_t *Bucket,
    GrB_Matrix C,
    GrB_Matrix M,
    GrB_Matrix A,
    GrB_Matrix B,
    int sz
)
{

    T_A *Ax = (T_A*)A->x;
    T_B *Bx = (T_B*)B->x;
    T_C *Cx = (T_C*)C->x;
    int64_t *Ci = C->i;
    int64_t *Mi = M->i;
    int64_t *Ai = A->i;
    int64_t *Bi = B->i;
    int64_t *Ap = A->p;
    int64_t *Bp = B->p;


    // zombie count
    int zc = 0;

    int64_t pair_id;

    // set thread ID
    int tid_global = threadIdx.x+ blockDim.x* blockIdx.x;
    int tid = threadIdx.x;

    int b = blockIdx.x ;

    // total items to be inspected
    int64_t nnzA = 0;
    int64_t nnzB = 0;
    int64_t n_intersect = 0;

    thread_block_tile<tile_sz> tile = tiled_partition<tile_sz>( this_thread_block());

    int parts = blockDim.x; //(n_intersect+ intersects_per_thread -1)/ intersects_per_thread; 

    // int has_zombies = 0 ;

    // Main loop over pairs 
    for (pair_id = start+ blockIdx.x; //warp per pair 
         pair_id < end;  
         pair_id += gridDim.x )
    {

         int64_t i = Mi[pair_id];
         int64_t j = Ci[pair_id] >> 4;

         int64_t xstart = Ap[i];
         int64_t xend   = Ap[i+1];
         nnzA = xend - xstart;

         int64_t ystart = Bp[j]; 
         int64_t yend   = Bp[j+1]; 
         nnzB = yend - ystart;

         n_intersect = GB_IMIN( xend -xstart, yend -ystart); 
    /* 
    if (threadIdx.x ==0 ) {
      printf("block %d  doing dot %lld  i,j= %lld,%lld\n", blockIdx.x, pair_id, i, j);
    }
    */
    //we want more than one intersection per thread
    int64_t nxy = nnzA + nnzB;

    int work_per_thread = (nxy +parts -1)/parts;
    int diag = GB_IMIN( work_per_thread*tid, nxy);
    int diag_end = GB_IMIN( diag + work_per_thread, nxy);
    //printf(" thd%d parts = %u wpt = %u diag, diag_end  = %u,%u\n",tid, parts, work_per_thread, diag, diag_end); 

    int x_min = GB_IMAX( (int)(diag - nnzB), 0);
    int x_max = GB_IMIN( diag, nnzA);

    //printf("start thd%u x_min = %u x_max = %u\n", tid_global, x_min,x_max);
    while ( x_min < x_max) { //binary search for correct diag break
      int pivot = (x_min +x_max)/2;
      if ( Ai[pivot + xstart] < Bi[ diag -pivot -1 + ystart]) {
         x_min = pivot +1;
      }
      else {
         x_max = pivot;
      }
    }
    int xcoord = x_min;
    int ycoord = diag -x_min -1;
    if (( diag > 0) &&(diag < (nnzA+nnzB)) && (Ai[xcoord+xstart] == Bi[ycoord+ystart]) ) { 
       diag--; //adjust for intersection incrementing both pointers 
    }
    // two start points are known now
    int tx_start = xcoord +xstart;
    int ty_start = diag -xcoord +ystart; 

    //if (x_start != y_start)
    //   printf("start thd%u  xs,ys = %i,%i\n", tid_global, x_start, y_start);

    x_min = GB_IMAX( (int)(diag_end - nnzB), 0);
    x_max = GB_IMIN( diag_end, nnzA);

    while ( x_min < x_max) {
       int pivot = (x_min +x_max)/2;
       //printf("thd%u pre_sw piv=%u diag_e = %u  xmin,xmax=%u,%u\n", tid_global, pivot, diag_end,x_min, x_max);
       if ( Ai[pivot+ xstart] < Bi[ diag_end -pivot -1 +ystart]) {
          x_min = pivot +1;
       }
       else {
          x_max = pivot;
       }
       //printf("thd%u piv=%u xmin,xmax = %u,%u\n", tid_global, pivot, x_min, x_max);
    }
    xcoord = x_min;
    ycoord = diag_end -x_min -1;
    if ( (diag_end < (nnzA +nnzB)) && (Ai[xcoord +xstart] == Bi[ycoord + ystart]) ) { 
        diag--; //adjust for intersection incrementing both pointers  
    }
    // two end points are known now
    int tx_end = xcoord +xstart; 
    int ty_end = diag_end - xcoord + ystart; 

    T_A aki;
    T_B bkj;
    T_Z cij = GB_IDENTITY ;

    // TODO PLUS_PAIR_INT64, FP32, FP64: no need for cij_exists.
    // just check if cij > 0

    int cij_exists  = 0 ;
    //printf(" thd%u has init value %f\n",tid, cij);

    //merge-path dot product
    int k = tx_start;
    int l = ty_start;
    while ( k < tx_end && l < ty_end )
    {
       if (Ai [k] == Bi [l])
       {
          GB_GETA ( aki=(T_Z)Ax[k] ) ;
          GB_GETB ( bkj=(T_Z)Bx[l] ) ;
          if (cij_exists)
          {
            T_Z t = GB_MULT( (T_Z)aki, (T_Z)bkj );
            GB_ADD_F (cij, t ) ;
          //printf("  thd%d ix at %lld   cij += %d * %d \n", tid_global, Ai[k], aki, bkj);
          }
          else
          {
            cij_exists = 1 ;
            cij = GB_MULT ( (T_Z)aki, (T_Z)bkj ) ;
          //printf("  thd%d ix at %lld   cij = %d * %d \n", tid_global, Ai[k], Ax[k], Bx[l]);
          }
          // TODO check terminal condition
          k+= 1;
          l+= 1;
          //printf(" block%u work value = %d, exists = %d\n", b, cij, cij_exists);
       }
       else
       {
            k += ( Ai[k] < Bi[l] ) ;
            l += ( Ai[k] > Bi[l] ) ;
       }
    }

    //tile.sync( ) ;
    //--------------------------------------------------------------------------
    // reduce sum per-thread values to a single scalar, get OR of flag
    //--------------------------------------------------------------------------
    /*
    if (tid == 0)
    {
        printf ("reduce %d : %d exists = %d\n", b,  cij, cij_exists) ;
    }
    __syncthreads();
    */

    // Do vote here for control.
    cij_exists  = tile.any( cij_exists);
    //tile.sync();

    if (cij_exists)
    {
       cij = GB_reduce_sum<T_Z, tile_sz>( tile, cij );
       
    }
    // else has_zombies = 1;


    //__syncthreads();
    //tile.sync( );
    // write result for this block to global mem
    if (tid == 0)
    {
        //printf ("final %d : %d exists = %d\n", b,  cij, cij_exists) ;
        if (cij_exists)
        {
           //printf(" cij = %d\n", cij);
           GB_PUTC ( Cx[pair_id]=(T_C)cij ) ;
           GB_PUTC ( Ci[pair_id]=i ) ;
        }
        else
        {
           //printf(" dot %d is a zombie\n", pair_id);
           zc++;
           GB_PUTC ( Ci[pair_id]=GB_FLIP (i) ) ;
        }
    }
    //__syncthreads(); 
  }

//--------------------------------------------------------------------------

  if( tid ==0 && zc > 0)
  {
      //printf("warp %d zombie count = %d\n", blockIdx.x, zc);
      atomicAdd( (unsigned long long int*)&(C->nzombies), (unsigned long long int)zc);
      //printf(" Czombie = %lld\n",C->nzombies);
  }

  //__syncthreads();

}

