#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// spGEMM_very_sparse_sparse.cu 
//------------------------------------------------------------------------------

// The spGEM_vssp CUDA kernel produces the semi-ring product of two
// sparse matrices of types T_A and T_B and common index space size n, to a  
// output matrix of type T_C. The matrices are sparse, with different numbers
// of non-zeros and different sparsity patterns. 
// ie. we want to produce C = A'*B in the sense of the given semi-ring.

// This version uses a binary-search algorithm, when the sizes nnzA and nnzB
// are far apart in size, neither is very spare nor dense, for any size of N.

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x. s= 32 with a variable number
// of active threads = min( min(nzA, nzB), 32) 

// Thus, each t in threadblock b owns a part of the set of pairs in the 
// sparse-sparse bucket of work. The job for each pair of vectors is to find 
// the intersection of the index sets Ai and Bi, perform the semi-ring dot 
// product on those items in the intersection, and finally
// on exit write it to Cx [pair].

//  int64_t start          <- start of vector pairs for this kernel
//  int64_t end            <- end of vector pairs for this kernel
//  int64_t *Bucket        <- array of pair indices for all kernels 
//  GrB_Matrix C         <- result matrix 
//  GrB_Matrix M         <- mask matrix
//  GrB_Matrix A         <- input matrix A
//  GrB_Matrix B         <- input matrix B

#include <limits>
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include "mySemiRing.h"
#include "matrix.h"

// Using tile size fixed at compile time, we don't need shared memory
#define tile_sz 32 

using namespace cooperative_groups;

template< typename T, int warpSize >
__device__ T reduce_sum(thread_block_tile<warpSize> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        val += g.shfl_down(val,i) ;
    }
    return val; // note: only thread 0 will return full sum
}

#define intersects_per_thread 8

template< typename T_C, typename T_A, typename T_B, typename T_X, typename T_Y, typename T_Z>
__global__ void AxB_dot3_phase3_vssp
(
    int64_t start,
    int64_t end,
    int64_t *Bucket,
    GrB_Matrix C,
    GrB_Matrix M,
    GrB_Matrix A,
    GrB_Matrix B,
    int sz
)
{
   // Typed pointers to access data in A,B,C
   T_A *Ax = (T_A*)A->x;
   T_B *Bx = (T_B*)B->x;
   T_C *Cx = (T_C*)C->x;
   int64_t *Ci = C->i;
   int64_t *Mi = M->i;
   int64_t *Ai = A->i;
   int64_t *Bi = B->i;
   int64_t *Ap = A->p;
   int64_t *Bp = B->p;

   // sz = expected non-zeros per dot 
   int m = 256/sz;
   int nvecs = end - start;
   int dpt = nvecs/(gridDim.x*32);
   
   int dots = (nvecs +dpt -1)/dpt; 

   // zombie count
   int zc = 0;
   int64_t pair_id, im;

   // set thread ID
   unsigned int tid_global = threadIdx.x+ blockDim.x* blockIdx.x;
   unsigned int tid = threadIdx.x;

   unsigned long int b = blockIdx.x ;

   // Main loop over pairs 
   for (pair_id = start+ tid_global, im = 0; 
        pair_id < end && im < m;  
        pair_id += gridDim.x*blockDim.x, ++im){

        int64_t i = Mi[pair_id];
        int64_t j = Ci[pair_id] >> 4;

        if( j < 0) //Pre-zombie
        {
            zc++;
            continue;
        }

        int64_t pA      = Ap[i];
        int64_t pA_end  = Ap[i+1];
        int64_t nnzA = pA_end - pA;

        int64_t pB      = B->p[j]; 
        int64_t pB_end  = B->p[j+1]; 
        int64_t nnzB = pB_end - pB;

        //Search for each nonzero in the smaller vector to find intersection 
        bool cij_exists = false;

        T_A aki;
        T_B bkj;
        T_Z cij;

        if (nnzA <= nnzB) {
            //----------------------------------------------------------------------
            // A(:,i) is very sparse compared to B(:,j)
            //----------------------------------------------------------------------

            while (pA < pA_end && pB < pB_end)
            {
                int64_t ia = Ai [pA] ;
                int64_t ib = Bi [pB] ;
                if (ia < ib)
                { 
                    // A(ia,i) appears before B(ib,j)
                    pA++ ;
                }
                else if (ib < ia)
                { 
                    // B(ib,j) appears before A(ia,i)
                    // discard all entries B(ib:ia-1,j)
                    int64_t pleft = pB + 1 ;
                    int64_t pright = pB_end - 1 ;
                    GB_TRIM_BINARY_SEARCH (ia, Bi, pleft, pright) ;
                    //ASSERT (pleft > pB) ;
                    pB = pleft ;
                }
                else // ia == ib == k
                { 
                    // A(k,i) and B(k,j) are the next entries to merge
                    #if defined ( GB_PHASE_1_OF_2 )
                    cij_exists = true ;
                    break ;
                    #else
                    GB_DOT_MERGE ;
                    //GB_DOT_TERMINAL (cij) ;         // break if cij == terminal
                    pA++ ;
                    pB++ ;
                    #endif
                }
            }
        }
        else {
            //----------------------------------------------------------------------
            // B(:,j) is very sparse compared to A(:,i)
            //----------------------------------------------------------------------

            while (pA < pA_end && pB < pB_end)
            {
                int64_t ia = Ai [pA] ;
                int64_t ib = Bi [pB] ;
                if (ia < ib)
                { 
                    // A(ia,i) appears before B(ib,j)
                    // discard all entries A(ia:ib-1,i)
                    int64_t pleft = pA + 1 ;
                    int64_t pright = pA_end - 1 ;
                    GB_TRIM_BINARY_SEARCH (ib, Ai, pleft, pright) ;
                    //ASSERT (pleft > pA) ;
                    pA = pleft ;
                }
                else if (ib < ia)
                { 
                    // B(ib,j) appears before A(ia,i)
                    pB++ ;
                }
                else // ia == ib == k
                { 
                    // A(k,i) and B(k,j) are the next entries to merge
                    #if defined ( GB_PHASE_1_OF_2 )
                    cij_exists = true ;
                    break ;
                    #else
                    GB_DOT_MERGE ;
                    //GB_DOT_TERMINAL (cij) ;         // break if cij == terminal
                    pA++ ;
                    pB++ ;
                    #endif
                }
            }

        }
        if ( cij_exists){
           GB_PUTC ( Ci[pair_id]=i ) ;
           GB_PUTC ( Cx[pair_id]=(T_C)cij ) ;
        }
        else {
           zc++; 
           //printf(" %lld, %lld is zombie %d!\n",i,j,zc);
           GB_PUTC( Ci[pair_id] = GB_FLIP( i ) ) ;
        }


    }

    //--------------------------------------------------------------------------
    // reduce sum per-thread values to a single scalar
    //--------------------------------------------------------------------------
    thread_block_tile<tile_sz> tile = tiled_partition<tile_sz>( this_thread_block());
    zc = reduce_sum<int,tile_sz>(tile, zc);

    if( threadIdx.x ==0) {
      //printf("warp %d zombie count = %d\n", blockIdx.x, zc);
      atomicAdd( (unsigned long long int*)&(C->nzombies), (unsigned long long int)zc);
      //printf(" Czombie = %lld\n",C->nzombies);
    }

}

