#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// denseDotProduct.cu 
//------------------------------------------------------------------------------

// The denseDotProduct CUDA kernel produces the semi-ring dot product of two
// vectors of types T1 and T2 and common size n, to a vector odata of type T3.
// ie. we want to produce dot(x,y) in the sense of the given semi-ring.

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x.
// Each threadblock owns s*8 contiguous items in the input data.

// Thus, threadblock b owns g_idata [b*s*8 ... min(n,(b+1)*s*8-1)].  It's job
// is to reduce this data to a scalar, and write it to g_odata [b].

#include <limits>
#include "mySemiRing.h"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

template< typename T3, int tile_sz>
__inline__ __device__ 
T3 warp_ReduceSum(thread_block_tile<tile_sz> g, T3 val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        T3 fold = g.shfl_down( val, i);
        val = ADD( val, fold );
    }
    return val; // note: only thread 0 will return full sum
}

template<typename T3, int warpSize>
__inline__ __device__
T3 block_ReduceSum(thread_block g, T3 val)
{
  static __shared__ T3 shared[warpSize]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;
  thread_block_tile<warpSize> tile = tiled_partition<warpSize>(g);

  // Each warp performs partial reduction
  val = warp_ReduceSum<T3,warpSize>(tile, val);    

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : (T3)MONOID_IDENTITY3;

  
  if (wid==0) val = warp_ReduceSum<T3,warpSize>(tile,val); //Final reduce within first warp

  return val;
}

template< typename T1, typename T2, typename T3>
__global__ void denseDotProduct
(
    T1 *g_xdata,     // array of size n, type T1
    T2 *g_ydata,     // array of size n, type T2
    T3 *g_odata,       // array of size grid.x, type T3
    unsigned int n
)
{
    // set thread ID
    unsigned int tid = threadIdx.x ;

    // this threadblock b owns g_idata [block_start ... block_end-1]
    unsigned long int s = blockDim.x ;
    unsigned long int b = blockIdx.x ;
    unsigned long int block_start = b * s * 8 ;
    unsigned long int block_end   = (b + 1) * s * 8 ;

    /*
    if (tid == 0)
    {
        printf ("block %d: [%lu ... %ld]\n", b, block_start, block_end-1) ;
    }
    */

    /*
    if (tid == 0 && b == 0)
    {
        printf ("type is size %d\n", sizeof (T)) ;
        for (int k = 0 ; k < n ; k++) printf ("%4d: %g\n", k, (double) g_idata [k]) ;
        printf ("\n") ;
    }
    */

    // each thread tid reduces its result into sum 
    T3 sum;

    // nothing to do
    if (block_start > block_end) { return ; }

    // convert global data pointer to the local pointer of this block
    T1 *xdata = g_xdata + block_start ;
    T2 *ydata = g_ydata + block_start ;

    T1 x0, x1, x2, x3, x4, x5, x6, x7 ;
    T2 y0, y1, y2, y3, y4, y5, y6, y7 ;

    if (block_end <= n)
    {
        // unrolling 8
        x0 = xdata [tid] ;
        x1 = xdata [tid +     s] ;
        x2 = xdata [tid + 2 * s] ;
        x3 = xdata [tid + 3 * s] ;
        x4 = xdata [tid + 4 * s] ;
        x5 = xdata [tid + 5 * s] ;
        x6 = xdata [tid + 6 * s] ;
        x7 = xdata [tid + 7 * s] ;

        y0 = ydata [tid] ;
        y1 = ydata [tid +     s] ;
        y2 = ydata [tid + 2 * s] ;
        y3 = ydata [tid + 3 * s] ;
        y4 = ydata [tid + 4 * s] ;
        y5 = ydata [tid + 5 * s] ;
        y6 = ydata [tid + 6 * s] ;
        y7 = ydata [tid + 7 * s] ;
        /*
        if (b == 0)
        {
            printf ("block zero: here is tid %2d : %g %g %g %g %g %g %g %g \n", tid,
                (double) x0, (double) x1, (double) x2, (double) x3,
                (double) x4, (double) x5, (double) x6, (double) x7) ;
        }
        */

    }
    else
    {
        // the last block has size less than 8*s
        #define XDATA(i) ((i < lastblocksize) ? xdata [i] : MONOID_IDENTITY1)
        #define YDATA(i) ((i < lastblocksize) ? ydata [i] : MONOID_IDENTITY2)
        int lastblocksize = n - block_start ;
        x0 = XDATA (tid) ;
        x1 = XDATA (tid +     s) ;
        x2 = XDATA (tid + 2 * s) ;
        x3 = XDATA (tid + 3 * s) ;
        x4 = XDATA (tid + 4 * s) ;
        x5 = XDATA (tid + 5 * s) ;
        x6 = XDATA (tid + 6 * s) ;
        x7 = XDATA (tid + 7 * s) ;

        y0 = YDATA (tid) ;
        y1 = YDATA (tid +     s) ;
        y2 = YDATA (tid + 2 * s) ;
        y3 = YDATA (tid + 3 * s) ;
        y4 = YDATA (tid + 4 * s) ;
        y5 = YDATA (tid + 5 * s) ;
        y6 = YDATA (tid + 6 * s) ;
        y7 = YDATA (tid + 7 * s) ;
    }

    //work [tid] = mul(x0,y0) + mul(x1,y1) + mul(x2,y2) + mul(x3,y3)
    //               + mul(x4,y4) + mul(x5,y5) + mul(x6,y6)+ mul(x7,y7) ;
          sum  = ADD( MUL(x0,y0) , ADD( MUL(x1,y1) , ADD( MUL(x2,y2), 
                 ADD( MUL(x3,y3) , ADD( MUL(x4,y4) , ADD( MUL(x5,y5), 
                 ADD( MUL(x6,y6) , MUL(x7,y7)))))))) ;

        /*
        if (b == 0)
        {
            printf ("block zero: still is tid %2d : %g %g %g %g %g %g %g %g \n", tid,
                (double) x0, (double) x1, (double) x2, (double) x3,
                (double) x4, (double) x5, (double) x6, (double) x7) ;
        }

        if (b == 0)
        {
            printf ("block zero: here is tid %d result %g  is %g\n",
            tid, sum,
            (double) (x0 + x1 + x2 + x3 + x4 + x5 + x6 + x7)) ;
        }
        */

    __syncthreads ( ) ;

    //--------------------------------------------------------------------------
    // reduce per-thread sums to a single scalar
    //--------------------------------------------------------------------------

    sum = block_ReduceSum<T3, 32>( this_thread_block(), sum); 

    // write result for this block to global mem
    if (tid == 0)
    {
        printf ("final %d : %g\n", b, (T3) sum) ;
        g_odata [b] = sum ;
    }
}

