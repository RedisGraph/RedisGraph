#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// reduceUnrolled.cu
//------------------------------------------------------------------------------

// The reduceUnrolled CUDA kernel reduces an array g_idata of size n, of any
// type T, to an array g_odata of size grid.x.  Each threadblock (blockIdx.x)
// reduces its portion of g_idata to a single scalar, g_odata [blockIdx.x].

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x.
// Each threadblock owns s*8 contiguous items in the input data.

// Thus, threadblock b owns g_idata [b*s*8 ... min(n,(b+1)*s*8-1)].  It's job
// is to reduce this data to a scalar, and write it to g_odata [b].

#include "mySemiRing.h"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

template< typename T, int tile_sz>
__inline__ __device__ 
T warp_ReduceSum( thread_block_tile<tile_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        T fold = g.shfl_down( val, i);
        //printf("thd%d   %d OP %d is %d\n", threadIdx.x, val, fold, OP( val, fold));
        val = OP( val, fold );
    }
    //if (threadIdx.x ==0) printf("thd%d single warp sum is %d\n", threadIdx.x,  val);
    return val; // note: only thread 0 will return full sum
}

template<typename T, int warpSize>
__inline__ __device__
T block_ReduceSum(thread_block g, T val)
{
  static __shared__ T shared[warpSize]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;
  thread_block_tile<warpSize> tile = tiled_partition<warpSize>( g );

  // Each warp performs partial reduction
  val = warp_ReduceSum<T, warpSize>( tile, val);    

  // Wait for all partial reductions
  if (lane==0) { 
     //printf("thd%d warp%d sum is %d\n", threadIdx.x, wid, val);
     shared[wid]=val; // Write reduced value to shared memory
     //printf("thd%d stored warp %d sum %d\n", threadIdx.x, wid, val);
  }
  __syncthreads();              // Wait for all partial reductions

  if (wid > 0 || gridDim.x == 1 ) return val;
  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : MONOID_IDENTITY;
  //printf("thd%d warp loaded val = %d\n", threadIdx.x, lane, val);

  
  if (wid==0) val = warp_ReduceSum<T, warpSize>( tile, val); //Final reduce within first warp

  return val;
}

template< typename T>
__global__ void reduceWarp
(
    T *g_idata,     // array of size n
    T *g_odata,     // array of size grid.x
    unsigned int N
)
{
    // set thread ID
    unsigned int tid = threadIdx.x ;

    // each thread tid reduces its result into sum
    T sum = (T) MONOID_IDENTITY;

    for(int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < N; 
        i += blockDim.x * gridDim.x) {
        sum = OP( sum, g_idata[i]);
    }
    //printf("thd%d  sum is %d\n", threadIdx.x + blockDim.x*blockIdx.x, sum);
    __syncthreads();
    //--------------------------------------------------------------------------
    // reduce work [0..s-1] to a single scalar
    //--------------------------------------------------------------------------
    // this assumes blockDim is a multiple of 32
    sum = block_ReduceSum<T , 32>( this_thread_block(), sum); 

    // write result for this block to global mem
    if (tid == 0)
    {
        // printf ("final %d : %g\n", b, (double) work [0]) ;
        g_odata [blockIdx.x] = sum ;
    }
}

