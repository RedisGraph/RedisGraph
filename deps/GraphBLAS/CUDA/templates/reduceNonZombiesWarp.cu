#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// reduceUnrolled.cu
//------------------------------------------------------------------------------

// The reduceUnrolled CUDA kernel reduces an array g_idata of size n, of any
// type T, to an array g_odata of size grid.x.  Each threadblock (blockIdx.x)
// reduces its portion of g_idata to a single scalar, g_odata [blockIdx.x].

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x.
// Each threadblock owns s*8 contiguous items in the input data.

// Thus, threadblock b owns g_idata [b*s*8 ... min(n,(b+1)*s*8-1)].  It's job
// is to reduce this data to a scalar, and write it to g_odata [b].

#define GB_KERNEL
#include <limits>
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include "mySemiRing.h"

using namespace cooperative_groups;

template< typename T, int tile_sz>
__inline__ __device__ 
T warp_ReduceSum( thread_block_tile<tile_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        T fold = g.shfl_down( val, i);
        //printf("thd%d   %d OP %d is %d\n", threadIdx.x, val, fold, OP( val, fold));
        val = GB_ADD( val, fold );
    }
    //if (threadIdx.x ==0) printf("thd%d single warp sum is %d\n", threadIdx.x,  val);
    return val; // note: only thread 0 will return full sum
}

template<typename T, int warpSize>
__inline__ __device__
T block_ReduceSum(thread_block g, T val)
{
  static __shared__ T shared[warpSize]; // Shared mem for 32 partial sums
  int lane = threadIdx.x & 31 ; // % warpSize;
  int wid  = threadIdx.x >> 5 ; // / warpSize;
  thread_block_tile<warpSize> tile = tiled_partition<warpSize>( g );

  // Each warp performs partial reduction
  val = warp_ReduceSum<T, warpSize>( tile, val);    

  // Wait for all partial reductions
  if (lane==0) { 
     //printf("thd%d warp%d sum is %d\n", threadIdx.x, wid, val);
     shared[wid] = val; // Write reduced value to shared memory
     //printf("thd%d stored warp%d sum %d\n", threadIdx.x, wid, val);
  }
  __syncthreads();              // Wait for all partial reductions

  if (wid > 0 ) return val;
  //read from shared memory only if that warp existed
  else { 
    val = (threadIdx.x < (blockDim.x / warpSize) ) ? shared[lane] : GB_IDENTITY ;
    //if (lane < (blockDim.x/ warpSize) ) printf("thd%d warp%d loaded val = %d\n", threadIdx.x, lane, val);
    val = warp_ReduceSum<T, warpSize>( tile, val); //Final reduce within first warp
  }

  return val;
}

template< typename T>
__global__ void reduceNonZombiesWarp
(
    int64_t *index,  // array of size n
    T *g_idata,      // array of size n
    T *g_odata,      // array of size grid.x
    unsigned int N
)
{
    // set thread ID
    int tid = threadIdx.x ;

    // each thread tid reduces its result into sum
    T sum = (T) GB_IDENTITY;

    for(int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < N; 
        i += blockDim.x * gridDim.x) {
        if ( index[i] < 0) continue;
        T fold = g_idata[i];
        sum = GB_ADD( sum, fold );
    }
    //printf("thd%d  sum is %d\n", threadIdx.x + blockDim.x*blockIdx.x, sum);
    __syncthreads();
    //--------------------------------------------------------------------------
    // reduce work [0..s-1] to a single scalar
    //--------------------------------------------------------------------------
    // this assumes blockDim is a multiple of 32
    sum = block_ReduceSum< T, 32 >( this_thread_block(), sum) ; 

    // write result for this block to global mem
    if (tid == 0)
    {
        g_odata [blockIdx.x] = sum ;
    }
}

