#include "hip/hip_runtime.h"

//------------------------------------------------------------------------------
// AxB_dot3_phase3_dndn.cu 
//------------------------------------------------------------------------------

// This CUDA kernel produces the semi-ring product of two
// sparse matrices of types T_A and T_B and common index space size n, to a  
// output matrix of type T_C. The matrices are sparse, with different numbers
// of non-zeros and different sparsity patterns. 
// ie. we want to produce C = A'*B in the sense of the given semi-ring.

// This version uses a simple warp-based dense dot product algorithm, when the
// vectors coming from both A and B are dense, for any size of N.

// Both the grid and block are 1D, so blockDim.x is the # threads in a
// threadblock, and the # of threadblocks is grid.x

// Let b = blockIdx.x, and let s be blockDim.x. s= 32 with a variable number
// of active threads = min( min(nzA, nzB), 32) 

// Thus, threadblock b owns a semi-ring dot product on a pair of vectors. 
// The work is to load the data, do the multiply and add work and finally 
// reduce this data to a scalar, and write it to Cx[pair].

//  int64_t start          <- start of vector pairs for this kernel
//  int64_t end            <- end of vector pairs for this kernel
//  int64_t *Bucket        <- array of pair indices for all kernels 
//  GrB_Matrix C           <- result matrix 
//  GrB_Matrix M           <- mask matrix
//  GrB_Matrix A           <- input matrix A
//  GrB_Matrix B           <- input matrix B
//  int sz                 <- size parameter (not used) 

#include <limits>
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include "matrix.h"
#include "mySemiRing.h"

// Using tile size fixed at compile time, we don't need shared memory
#define tile_sz 32 

using namespace cooperative_groups;

template< typename T, int warp_sz>
__inline__ __device__ T warp_ReduceSum(thread_block_tile<warp_sz> g, T val)
{
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        T next = g.shfl_down( val, i) ;
        val = GB_ADD( val, next ); 
    }
    return val; // note: only thread 0 will return full sum
}

template<typename T, int warpSize >
__inline__ __device__
T block_ReduceSum(thread_block g, T val, T Ident)
{
  static __shared__ T shared[warpSize]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;
  thread_block_tile<warpSize> tile = tiled_partition<warpSize>(g);

  // Each warp performs partial reduction
  val = warp_ReduceSum< T, warpSize>(tile, val);    

  if (lane==0) shared[wid] = val; // Write reduced value to shared memory

  //tile.sync();                    // Wait for all partial reductions

  if (wid > 0 || gridDim.x == 1 ) return val;

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] :  Ident  ;

  if (wid==0) val = warp_ReduceSum< T, warpSize>(tile,val); //Final reduce within first warp

  return val;
}


template< typename T_C, typename T_A, typename T_B, typename T_X, typename T_Y, typename T_Z>
__global__ void AxB_dot3_phase3_dndn 
(
    int64_t start,
    int64_t end,
    int64_t *Bucket,
    GrB_Matrix C,
    GrB_Matrix M,
    GrB_Matrix A,
    GrB_Matrix B,
    int sz
)
{

    T_A *Ax = (T_A*)A->x;
    T_B *Bx = (T_B*)B->x;
    T_C *Cx = (T_C*)C->x;
    int64_t *Mi = M->i;
    int64_t *Ci = C->i;
    int64_t *Ap = A->p;
    int64_t *Bp = B->p;

    // zombie count
    int zc = 0;
    int64_t pair_id;

    // total items to be inspected
    int64_t nnzA = 0;
    int64_t nnzB = 0;
    int s = blockDim.x;

    // Main loop over pairs 
    for (pair_id = start + blockIdx.x; //warp per pair 
         pair_id < end;  
         pair_id += gridDim.x ){

         int64_t i = Mi[pair_id];
         int64_t j = Ci[pair_id] >> 4;

         int64_t pA = Ap[i];
         int64_t xend   = Ap[i+1];
         nnzA = xend - pA;

         int64_t pB = Bp[j]; 
         int64_t yend   = Bp[j+1]; 
         nnzB = yend - pB;

    /*
    if (threadIdx.x == 0 ){
        printf(" i,j = %d,%d  nnz= %d xstart,end = %d,%d  ystart,end = %d,%d\n",
            (int)i,(int)j,  (int)nnzA, (int)xstart,(int)xend, (int)ystart, (int)yend);
    }
    __syncthreads();                                          
    */

    
    // convert global data pointer to the local pointer of this block
    T_A  aki; // *xdata = &Ax[xstart]; 
    T_B  bkj; // *ydata = &Bx[ystart];
    T_Z  cij;

    GB_GETA ( aki=(T_Z)Ax[pA+threadIdx.x] ) ;             // aki = A(0,i)
    GB_GETB ( bkj=(T_Z)Bx[pB+threadIdx.x] ) ;             // bkj = B(0,j)
    GB_C_MULT ( cij, aki, bkj ) ;                        // cij = aki * bkj

    for ( int tid = threadIdx.x + s; tid < nnzA; tid+= s) { 
          // cij += A(k,i) * B(k,j)
          // GB_DOT_TERMINAL ( cij ) ;             // break if cij == terminal
          GB_GETA ( aki=(T_Z)Ax[pA+tid] ) ;         // aki = A(k,i)
          GB_GETB ( bkj=(T_Z)Bx[pB+tid] ) ;        // bkj = B(k,j)
          GB_MULTADD ( cij, aki, bkj ) ;        // cij += aki * bkj
    }


    //--------------------------------------------------------------------------
    // reduce per-thread sums to a single scalar
    //--------------------------------------------------------------------------
    thread_block_tile<32> tile = tiled_partition<32>( this_thread_block() );
    cij = warp_ReduceSum<T_Z, 32> ( tile, cij);

    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
       //printf("tid: %d final sum after reduce = %d\n", threadIdx.x, sum);
       GB_PUTC( Cx[pair_id]=(T_C)cij ) ;
       GB_PUTC( Ci[pair_id]=i ) ;
    }
    //__syncthreads ( ) ;
  }

}

