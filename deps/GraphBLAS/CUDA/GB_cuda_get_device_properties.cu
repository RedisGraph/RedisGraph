//------------------------------------------------------------------------------
// GB_cuda_get_device_properties.cu: get the properties of a GPU
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"

//------------------------------------------------------------------------------
// GB_cuda_get_device: get the current GPU
//------------------------------------------------------------------------------

bool GB_cuda_get_device (int &device)
{
    if (&device == NULL)
    {
        // invalid inputs
        return (false) ;
    }
    CHECK_CUDA_SIMPLE (hipGetDevice (&device)) ;
    return (true) ;
}

//------------------------------------------------------------------------------
// GB_cuda_set_device: set the current GPU
//------------------------------------------------------------------------------

bool GB_cuda_set_device (int device)
{
    if (device < 0)
    {
        // invalid inputs
        return (false) ;
    }
    CHECK_CUDA_SIMPLE (hipSetDevice (device)) ;
    return (true) ;
}

//------------------------------------------------------------------------------
// GB_cuda_get_device_properties: determine all properties of a single GPU
//------------------------------------------------------------------------------

bool GB_cuda_get_device_properties  // true if OK, false if failure
(
    int device,
    GB_cuda_device *prop
)
{

    //--------------------------------------------------------------------------
    // check inputs
    //--------------------------------------------------------------------------

    if (prop == NULL || device < 0)
    {
        // invalid inputs
        return (false) ;
    }

    // clear the GPU settings
    memset (prop, 0, sizeof (GB_cuda_device)) ;

    int old_device ;
    CHECK_CUDA_SIMPLE ( hipGetDevice( &old_device ) ) ;

    //--------------------------------------------------------------------------
    // get the properties
    //--------------------------------------------------------------------------

    int num_sms, compute_capability_major, compute_capability_minor ;
    size_t memfree, memtotal ;

    CHECK_CUDA_SIMPLE( hipDeviceGetAttribute (&num_sms,
                                         hipDeviceAttributeMultiprocessorCount,
                                         device) ) ;
    CHECK_CUDA_SIMPLE( hipDeviceGetAttribute (&compute_capability_major,
                                         hipDeviceAttributeComputeCapabilityMajor,
                                         device) ) ;
    CHECK_CUDA_SIMPLE( hipDeviceGetAttribute (&compute_capability_minor,
                                         hipDeviceAttributeComputeCapabilityMajor,
                                         device) ) ;

    CHECK_CUDA_SIMPLE ( hipSetDevice( device ) ) ;
    CHECK_CUDA_SIMPLE ( hipMemGetInfo( & memfree, &memtotal) ) ;
    CHECK_CUDA_SIMPLE ( hipSetDevice( old_device ) ) ;

    prop->total_global_memory = memtotal ;
    prop->number_of_sms = num_sms ;
    prop->compute_capability_major = compute_capability_major ;
    prop->compute_capability_minor = compute_capability_minor ;

    printf ("Device: %d: memory: %ld SMs: %d compute: %d.%d\n",
        device, prop->total_global_memory, prop->number_of_sms,
        prop->compute_capability_major, prop->compute_capability_minor) ;

    //--------------------------------------------------------------------------
    // return result
    //--------------------------------------------------------------------------

    return (true) ;
}

