//------------------------------------------------------------------------------
// GB_cuda_get_device_properties.cu: get the properties of a GPU
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"

bool GB_cuda_get_device ( int &device){
    bool goodreturn = false;
    if (&device == NULL)
    {
        // invalid inputs
        return (false) ;
    }

    CHECK_CUDA_SIMPLE ( hipGetDevice( &device ) ); 
    goodreturn = true;

    return goodreturn;

}

bool GB_cuda_set_device( int device) {
    bool goodreturn = false;
    if (device < 0)
    {
        // invalid inputs
        return (false) ;
    }

    CHECK_CUDA_SIMPLE ( hipSetDevice( device ) ); 
    goodreturn = true;

    return goodreturn;
}

bool GB_cuda_get_device_properties  // true if OK, false if failure
(
    int device,
    GB_cuda_device *prop
)
{

    //--------------------------------------------------------------------------
    // check inputs
    //--------------------------------------------------------------------------
    bool goodreturn = false;
    if (prop == NULL || device < 0)
    {
        // invalid inputs
        return (false) ;
    }

    int old_device;
    CHECK_CUDA_SIMPLE ( hipGetDevice( &old_device ) ) ; 


    //--------------------------------------------------------------------------
    // get the properties
    //--------------------------------------------------------------------------
    int num_sms;
    int compute_capability_major;
    int compute_capability_minor;
    size_t memfree, memtotal;

    CHECK_CUDA_SIMPLE( hipDeviceGetAttribute (&num_sms,
                                         hipDeviceAttributeMultiprocessorCount,
                                         device) );
    CHECK_CUDA_SIMPLE( hipDeviceGetAttribute (&compute_capability_major,
                                         hipDeviceAttributeComputeCapabilityMajor,
                                         device) );
    CHECK_CUDA_SIMPLE( hipDeviceGetAttribute (&compute_capability_minor,
                                         hipDeviceAttributeComputeCapabilityMajor,
                                         device) );

    CHECK_CUDA_SIMPLE ( hipSetDevice( device ) ); 
    CHECK_CUDA_SIMPLE ( hipMemGetInfo( & memfree, &memtotal) ) ;
    CHECK_CUDA_SIMPLE ( hipSetDevice( old_device ) ); 

    prop->total_global_memory = memtotal;
    prop->number_of_sms = num_sms;
    prop->compute_capability_major = compute_capability_major;
    prop->compute_capability_minor = compute_capability_minor;
    
    goodreturn = true;
    //--------------------------------------------------------------------------
    // return result
    //--------------------------------------------------------------------------

    return  goodreturn;
}

