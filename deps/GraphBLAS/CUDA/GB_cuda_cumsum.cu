//------------------------------------------------------------------------------
// GB_cuda_cumsum: cumlative sum of an array using GPU acceleration
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

// Compute the cumulative sum of an array count[0:n], of size n+1
// in pseudo-MATLAB notation:

//      k = sum (count [0:n-1] != 0) ;

//      count = cumsum ([0 count[0:n-1]]) ;

// That is, count [j] on input is overwritten with the value of
// sum (count [0..j-1]).  count [n] is implicitly zero on input.
// On output, count [n] is the total sum.

#include "GB_cuda.h"
#include <local_cub/device/device_scan.cuh>

GrB_Info GB_cuda_cumsum             // compute the cumulative sum of an array
(
    int64_t *restrict count,    // size n+1, input/output
    const int64_t n
)
{
    //--------------------------------------------------------------------------
    // check inputs
    //--------------------------------------------------------------------------

    ASSERT (count != NULL) ;
    ASSERT (n >= 0) ;

    //--------------------------------------------------------------------------
    // count = cumsum ([0 count[0:n-1]]) ;
    //--------------------------------------------------------------------------
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, count, count, (int)n);
    d_temp_storage  = GB_malloc_memory( temp_storage_bytes, 1);
    if ( d_temp_storage == NULL){
       return GrB_OUT_OF_MEMORY;
    } 

    // Run
    CubDebugExit(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, count, count, n));

    // Check for correctness (and display results, if specified)
    #ifdef GB_DEBUG
    int compare = CompareDeviceResults(h_reference, count, num_items, true, g_verbose);
    ASSERT( compare == 0);
    #endif

    // Cleanup
    GB_free_memory(d_temp_storage ) ; 

    return GrB_SUCCESS;
}





