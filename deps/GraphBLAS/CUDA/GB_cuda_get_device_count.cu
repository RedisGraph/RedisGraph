//------------------------------------------------------------------------------
// GB_cuda_get_device_count.cu: find out how many GPUs exist
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"

bool GB_cuda_get_device_count   // true if OK, false if failure
(
    int *gpu_count              // return # of GPUs in the system
)
{
    hipError_t err = hipGetDeviceCount (gpu_count) ;
    return (err == hipSuccess) ;
}

