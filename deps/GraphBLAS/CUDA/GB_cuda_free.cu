//------------------------------------------------------------------------------
// GB_cuda_free.cu: wrapper for hipFree
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"
#include "rmm/detail/cnmem.h"

void GB_cuda_free (void *p)     // standard free signature
{
    cnmemFree( p , NULL);
    //printf(" GPU %d freeing mem\n", device);
}

