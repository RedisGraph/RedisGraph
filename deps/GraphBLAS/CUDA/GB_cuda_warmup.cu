//------------------------------------------------------------------------------
// GB_cuda_warmup.cu: warmup the GPU
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"
/*
#include "rmm/include/rmm/mr/device/managed_memory_resource.hpp"
#include "rmm/include/rmm/mr/device/pool_memory_resource.hpp"
#include "rmm/include/rmm/mr/device/owning_wrapper.hpp"
#include "rmm/include/rmm/mr/device/default_memory_resource.hpp"
#include "rmm/include/rmm/mr/device/per_device_resource.hpp"
#include "rmm/include/rmm/mr/device/cnmem_managed_memory_resource.hpp"
*/
#include "rmm/detail/cnmem.h"

bool GB_cuda_warmup (int device)
{
    // allocate 'nothing' just to load the drivers.
    // No need to free the result.
    double gpu_memory_size = GB_Global_gpu_memorysize_get (device);

    printf ("warming up device %d memsize %g sms %d\n",
        device,
        gpu_memory_size, 
        GB_Global_gpu_sm_get (device)) ;


    //auto cuda_managed = std::make_shared<rmm::mr::managed_memory_resource>();
    //auto cuda = std::make_shared<rmm::mr::cuda_memory_resource>();
    //auto pool = rmm::mr::make_owning_wrapper<rmm::mr::pool_memory_resource>
    //            ( cuda_managed, gpu_memory_size/2, gpu_memory_size ) ;  

    std::vector<int> dev{0};
    cnmemDevice_t cnmem_device;
    memset(&cnmem_device, 0, sizeof(cnmem_device) ) ;
    cnmem_device.size = gpu_memory_size/2;
    if( device ==0)
    {
      cnmemInit(1, &cnmem_device, CNMEM_FLAGS_MANAGED);
    }

    //auto pool = std::make_shared<rmm::mr::cnmem_managed_memory_resource> ( gpu_memory_size/2 ) ;


    //rmm::mr::set_per_device_resource ( rmm::cuda_device_id{device}, 
    //                                 ( rmm::mr::device_memory_resource *)pool.get() ) ;
    
    //rmm::mr::set_default_resource ( pool.get() );
    //rmm::mr::set_current_device_resource ( pool.get() );

    //GB_Global_gpu_device_memory_resource_set( device, (void *)rmm::mr::get_current_device_resource() );

    void *p ;
    //hipError_t err = hipMalloc (&p, (size_t) 0) ;
    //p = rmm::mr::get_current_device_resource()->allocate(  256) ;
    //p = pool->allocate( 10) ;
    cnmemMalloc( &p,  256 , NULL);
    //rmm::mr::get_current_device_resource()->deallocate(p, 1);
    //pool->deallocate( p, 10);
    cnmemFree( p, NULL);

    printf ("GPU %d nice and toasty now, pool=%g\n", device, gpu_memory_size/2 ) ;

    // TODO check for jit cache? or in GB_init?

    return  true; //(err == hipSuccess) ;
}

