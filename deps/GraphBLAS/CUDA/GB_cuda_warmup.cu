//------------------------------------------------------------------------------
// GB_cuda_warmup.cu: warmup the GPU
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"

bool GB_cuda_warmup (int device)
{
    // allocate 'nothing' just to load the drivers.
    // No need to free the result.
    bool ok = GB_cuda_set_device( device );
    if (!ok)
    {
        printf ("invalid GPU: %d\n", device) ;
        return (false) ;
    }

    double gpu_memory_size = GB_Global_gpu_memorysize_get (device);

    printf ("warming up device %d memsize %g sms %d\n",
        device,
        gpu_memory_size, 
        GB_Global_gpu_sm_get (device)) ;

    size_t size = 0 ;
    void *p = GB_malloc_memory (1, 1, &size) ;
    if (p == NULL)
    {
        printf ("Hey!! where's da memory???\n") ;
        return (false) ;
    }
    printf ("oooo nice block of memory of size %lu\n", size) ;
    GB_free_memory ( &p, size) ;
    printf ("be free, block of memory of size %lu\n", size) ;

    printf ("good ol' hipMalloc just to be sure\n");
    hipMalloc ( &p, size ) ;
    if (p == NULL)
    {
        printf ("Hey!! where's da GPU???\n") ;
        return (false) ;
    }
    hipFree (p) ;

    printf ("GPU %d nice and toasty now\n", device) ;

    // TODO check for jit cache? or in GB_init?

    return  true; //(err == hipSuccess) ;
}

