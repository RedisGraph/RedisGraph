//------------------------------------------------------------------------------
// GB_cuda_calloc.cu: wrapper for hipMallocManaged and memset
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

#include "GB_cuda.h"

void *GB_cuda_calloc (size_t n, size_t size)   // standcard calloc signature
{

    // malloc the space
    void *p = GB_cuda_malloc (n * size) ;

    if (p == NULL)
    {
        // out of memory, or other CUDA error
        return (NULL) ;
    }

    // set the space to zero
    memset (p, 0, n * size) ;

    // return the result
    return (p) ;
}

