//------------------------------------------------------------------------------
// GB_AxB_dot3_cuda: compute C<M> = A'*B in parallel, on the GPU(s)
//------------------------------------------------------------------------------

// SPDX-License-Identifier: Apache-2.0
// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2019, All Rights Reserved.
// http://suitesparse.com   See GraphBLAS/Doc/License.txt for license.

//------------------------------------------------------------------------------

// This function only computes C<M>=A'*B on the GPUs.  The mask must be
// present, and not complemented.  The mask is always applied.

extern "C" 
{
  #include "GB_mxm.h"
}
#include "GB_cuda.h"


#include "templates/GB_jit_AxB_dot3_phase1.cu.jit"
#include "templates/GB_jit_AxB_dot3_phase2.cu.jit"
// the 5 kernels for the 5 buckets:
#include "templates/GB_jit_AxB_dot3_phase3_dndn.cu.jit"
#include "templates/GB_jit_AxB_dot3_phase3_vsvs.cu.jit"
#include "templates/GB_jit_AxB_dot3_phase3_vssp.cu.jit"
#include "templates/GB_jit_AxB_dot3_phase3_spdn.cu.jit"
#include "templates/GB_jit_AxB_dot3_phase3_mp.cu.jit"
#include "templates/GB_jit_AxB_dot3_phase3_warpix.cu.jit"
#include "templates/reduceNonZombiesWarp.cu.jit"

#include "GB_jit_launcher.h"


const std::vector<std::string> header_names ={};


#define GB_FREE_WORK                                                    \
{                                                                       \
    GB_cuda_free (Nanobuckets) ;    Nanobuckets = NULL ;                    \
    GB_cuda_free (Blockbucket) ;    Blockbucket = NULL ;                    \
    GB_cuda_free (Bucket);          Bucket      = NULL;                     \
    GB_cuda_free (Bucketp);         Bucketp     = NULL;                     \
    GB_cuda_free (offset);          offset      = NULL;                     \
}

#define GB_FREE_ALL                                                     \
{                                                                       \
    GB_FREE_WORK ;                                                      \
    GrB_Matrix_free (Chandle) ;                                         \
}

GrB_Info GB_AxB_dot3_cuda           // C<M> = A'*B using dot product method
(
    GrB_Matrix *Chandle,            // output matrix
    const GrB_Matrix M,             // mask matrix
    const bool Mask_struct,         // if true, use the only structure of M
    const GrB_Matrix A,             // input matrix
    const GrB_Matrix B,             // input matrix
    const GrB_Semiring semiring,    // semiring that defines C=A*B
    const bool flipxy,              // if true, do z=fmult(b,a) vs fmult(a,b)
    GB_Context Context
)
{

    //--------------------------------------------------------------------------
    // check inputs
    //--------------------------------------------------------------------------

    GrB_Info info ;
    ASSERT (Chandle != NULL) ;
    ASSERT (*Chandle == NULL) ;

    ASSERT_MATRIX_OK (M, "M for dot3 cuda A'*B", GB0) ;
    ASSERT_MATRIX_OK (A, "A for dot3 cuda A'*B", GB0) ;
    ASSERT_MATRIX_OK (B, "B for dot3 cuda A'*B", GB0) ;

    ASSERT (!GB_PENDING (M)) ;
    ASSERT (GB_JUMBLED_OK (M)) ;
    ASSERT (!GB_ZOMBIES (M)) ;

    ASSERT (!GB_PENDING (A)) ;
    ASSERT (!GB_JUMBLED (A)) ;
    ASSERT (!GB_ZOMBIES (A)) ;

    ASSERT (!GB_PENDING (B)) ;
    ASSERT (!GB_ZOMBIES (B)) ;
    ASSERT (!GB_JUMBLED (B)) ;

    ASSERT_SEMIRING_OK (semiring, "semiring for dot3 numeric A'*B", GB0) ;

    ASSERT (A->vlen == B->vlen) ;
    GBURBLE ("(GPU dot3) ") ;

    //--------------------------------------------------------------------------
    // initializations
    //--------------------------------------------------------------------------

    int ntasks = 0, number_of_sms = 0 ;
    int64_t *Nanobuckets = NULL, *Blockbucket = NULL ;
    int64_t *Bucket = NULL;
    int64_t *Bucketp = NULL;
    int64_t *offset = NULL;
    (*Chandle) = NULL ;

    // just in case M is jumbled and we don't handle it yet (TODO)
    GB_MATRIX_WAIT (M) ;
    ASSERT (!GB_JUMBLED (M)) ;

    int device = -1;

    hipSetDevice( 0 ) ;

    hipGetDevice(&device);

    //--------------------------------------------------------------------------
    // get M
    //--------------------------------------------------------------------------

    const int64_t *restrict Mp = M->p ;
    const int64_t *restrict Mh = M->h ;
    // const int64_t *restrict Mi = M->i ;
    // const GB_void *restrict Mx = M->x ;
    // const size_t msize = M->type->size ;
    const int64_t mvlen = M->vlen ;
    const int64_t mvdim = M->vdim ;
    const int64_t mnz = GB_NNZ (M) ;
    const int64_t mnvec = M->nvec ;
    const bool M_is_hyper = GB_IS_HYPERSPARSE( M ) ;

    const int64_t anz = GB_NNZ (A) ;
    const int64_t anvec = A->nvec ;

    const int64_t bnz = GB_NNZ (B) ;
    const int64_t bnvec = B->nvec ;

    //--------------------------------------------------------------------------
    // allocate C, the same size and # of entries as M
    //--------------------------------------------------------------------------

    // FUTURE: ctype need not be the op->ztype
    GrB_Type ctype = semiring->add->op->ztype ;
    int64_t cvlen = mvlen ;
    int64_t cvdim = mvdim ;
    int64_t cnz = mnz ;
    int64_t cnvec = mnvec ;

    // TODO tell GB_CREATE where to put the data: CPU or GPU (via
    // hipMemAdvise), but this works as-is.
    int sparsity = (M_is_hyper) ? GxB_HYPERSPARSE : GxB_SPARSE ;
    info = GB_new_bix (Chandle, // sparse or hyper (from M), new header
        ctype, cvlen, cvdim, GB_Ap_malloc, true,
        sparsity, false, M->hyper_switch, cnvec,
        cnz+1,  // add one to cnz for GB_cumsum of Cwork 
        true, Context) ;

    if (info != GrB_SUCCESS)
    { 
        // out of memory
        GB_FREE_ALL ;
        return (info) ;
    }

    GrB_Matrix C = (*Chandle) ;
    //int64_t *Citemp =  C->i ;        
    //auto *Cxtemp = C->x ;        
    //hipMalloc ((void**) &(C->i), cnz * sizeof( int64_t) ); 
    //hipMalloc ((void**) &(C->x), cnz * C->type->size ); 
    hipMemAdvise( C->i, cnz * sizeof ( int64_t), hipMemAdviseSetPreferredLocation, device); 
    hipMemAdvise( C->x, cnz * C->type->size , hipMemAdviseSetPreferredLocation, device); 

    int64_t *restrict Cp = M->p ;
    int64_t *restrict Ch = M->h ;
    // int64_t *restrict Ci = C->i ;
    // use C->i as workspace

    //--------------------------------------------------------------------------
    // copy Mp and Mh into C
    //--------------------------------------------------------------------------

    //hipMemcpy (Cp, Mp, (cnvec+1) * sizeof (int64_t), hipMemcpyDefault) ;
    if (M_is_hyper)
    { 
        //hipMemcpy (Ch, Mh, cnvec * sizeof (int64_t), hipMemcpyDefault) ;
    }
    C->magic = GB_MAGIC ;
    C->nvec_nonempty = M->nvec_nonempty ;
    C->nvec = M->nvec ;

    GBURBLE ("(GPU C created and copied from M) ") ;
    //--------------------------------------------------------------------------
    // stringify the semiring and the mask
    //--------------------------------------------------------------------------

    char semiring_name [GB_CUDA_STRLEN+2] ;
    char semiring_code [GB_CUDA_STRLEN+2] ;
    char mask_name [GB_CUDA_STRLEN+2] ;

    GB_cuda_stringify_semiring (semiring, flipxy,
        ctype, A->type, B->type, M->type, Mask_struct,  // matrix types
        true, semiring_name, semiring_code, mask_name) ;

    GBURBLE ("(GPU stringified) ") ;
    //--------------------------------------------------------------------------
    // construct the tasks for phase1 and phase2
    //--------------------------------------------------------------------------

    // on the CPU: nthreads = GB_nthreads (cnz, chunk, nthreads_max) ;
    // on the GPU:

    // # of threads in phase1 and phase2 kernel launches must be the same
    #define chunksize 128 
    #define SYMBOLIC_PHASE_NTHREADS 32 
    #define NBUCKETS (GB_BUCKET_MERGEPATH + 1)

    number_of_sms = GB_Global_gpu_sm_get (0) ;
    // C and M have cnz entries, so create ...
    //ntasks = ( (mnvec +7)/8   + SYMBOLIC_PHASE_NTHREADS -1 )/SYMBOLIC_PHASE_NTHREADS;
    ntasks =  ( mnz +chunksize -1)/chunksize;
    // Idea is to have each task work on a continguous block of columns of C
    ntasks = GB_IMIN( ntasks,  128*number_of_sms) ;    // ntasks will be grid.x

    GBURBLE ("(GPU mnz=%ld mnvec=%ld blockDim=32, nblock= %d) ", mnz, mnvec, ntasks ) ;

    std::cout<< "ntasks, nthreads = " <<ntasks<<","<<SYMBOLIC_PHASE_NTHREADS<<std::endl; 
    //--------------------------------------------------------------------------
    // phase1 and phase2: place each C(i,j) in a bucket
    //--------------------------------------------------------------------------

    hipMalloc ((void**) &Nanobuckets,
        NBUCKETS * SYMBOLIC_PHASE_NTHREADS * ntasks * sizeof (int64_t)) ;

    //Nanobuckets = (int64_t*)GB_cuda_malloc (
    //    NBUCKETS * SYMBOLIC_PHASE_NTHREADS * ntasks * sizeof (int64_t)) ;
    //hipMemAdvise( Nanobuckets, NBUCKETS * SYMBOLIC_PHASE_NTHREADS * ntasks
    //                           * sizeof ( int64_t), hipMemAdviseSetPreferredLocation, device); 
    /*
    */

    hipMalloc ((void**) &Blockbucket,
        NBUCKETS * ntasks* sizeof (int64_t) ) ;
    //Blockbucket = (int64_t*)GB_cuda_malloc ( NBUCKETS * ntasks* sizeof (int64_t) ) ;
    //hipMemAdvise( Blockbucket, NBUCKETS * ntasks
    //                           * sizeof ( int64_t), hipMemAdviseSetPreferredLocation, device); 
    /*
    */

    hipMalloc ((void**) &Bucket, cnz*sizeof(int64_t));
    //Bucket = (int64_t*)GB_cuda_malloc ( cnz*sizeof(int64_t) );
    //hipMemAdvise( Bucket, cnz * sizeof ( int64_t), hipMemAdviseSetPreferredLocation, device); 
    /*
    */

    //hipMallocManaged ((void**) &Bucketp, (NBUCKETS+1)*sizeof(int64_t)) ;
    Bucketp = (int64_t*)GB_cuda_malloc( (NBUCKETS+1)*sizeof(int64_t) ) ;
    hipMemAdvise( Bucketp, (NBUCKETS+1) * sizeof ( int64_t), hipMemAdviseSetPreferredLocation, hipCpuDeviceId); 
    hipMemAdvise( Bucketp, (NBUCKETS+1) * sizeof ( int64_t), hipMemAdviseSetAccessedBy, device); 

    //hipMallocManaged ((void**) &offset, (NBUCKETS)*sizeof(int64_t)) ;
    offset = (int64_t*)GB_cuda_malloc( (NBUCKETS)*sizeof(int64_t) ) ;
    hipMemAdvise( offset, NBUCKETS * sizeof ( int64_t), hipMemAdviseSetPreferredLocation, hipCpuDeviceId); 
    hipMemAdvise( offset, NBUCKETS * sizeof ( int64_t), hipMemAdviseSetAccessedBy, device); 

    memset( offset, 0, NBUCKETS * sizeof(int64_t) ); 
    
  /* 
    if (Blockbucket == NULL || Nanobuckets == NULL || Bucket == NULL || Bucketp == NULL )
    { 
        // out of memory
        GB_FREE_ALL ;
        return (GB_OUT_OF_MEMORY) ;
    }
    */
    

    //--------------------------------------------------------------------------
    // Pre-fetch arrays that will be used on the device
    //--------------------------------------------------------------------------

    
    //hipMemPrefetchAsync( Nanobuckets, NBUCKETS * SYMBOLIC_PHASE_NTHREADS
    //                     * ntasks * sizeof (int64_t), device, NULL) ;

    //hipMemPrefetchAsync( Blockbucket, NBUCKETS * ntasks 
    //                        * sizeof (int64_t), device, NULL) ;

    //hipMemPrefetchAsync( Bucket, cnz * sizeof (int64_t), device, NULL) ;
    

    /*
    
    //hipStream_t stream_data;
    //hipStreamCreate ( &stream_data);
    */
    /* 
    hipMemAdvise( M->p, (mnvec+1) * sizeof (int64_t), hipMemAdviseSetPreferredLocation, device) ;
    hipMemAdvise( M->i, mnz * sizeof ( int64_t), hipMemAdviseSetPreferredLocation, device); 
    hipMemAdvise( M->x, mnz * M->type->size, hipMemAdviseSetPreferredLocation,device) ;
    
    hipMemAdvise( M->p, (mnvec+1) * sizeof (int64_t), hipMemAdviseSetReadMostly, device) ;
    hipMemAdvise( M->i, mnz * sizeof (int64_t), hipMemAdviseSetReadMostly, device) ;
    hipMemAdvise( M->x, mnz * M->type->size, hipMemAdviseSetReadMostly,device) ;
    */

    hipMemPrefetchAsync( M->p, (mnvec+1) * sizeof (int64_t), device, NULL) ; //stream_data) ;
    hipMemPrefetchAsync( M->i, mnz * sizeof (int64_t), device, NULL ) ; //stream_data) ;
    hipMemPrefetchAsync( M->x, mnz * M->type->size, device, NULL ) ; //stream_data) ;
    /*
    hipMemAdvise( C->p, (mnvec+1) * sizeof (int64_t), hipMemAdviseSetReadMostly, device) ;
    hipMemAdvise( C->i, mnz * sizeof (int64_t), hipMemAdviseSetReadMostly, device) ;
    hipMemAdvise( C->x, mnz * C->type->size, hipMemAdviseSetReadMostly,device) ;
    */
    //hipMemPrefetchAsync( C->p, (mnvec+1) * sizeof (int64_t), device, NULL) ; //stream_data) ;
    hipMemPrefetchAsync( C->i, mnz * sizeof (int64_t), device, NULL ); //stream_data) ;
    hipMemPrefetchAsync( C->x, mnz * C->type->size, device, NULL ); //stream_data) ;
    
    /*
    hipMemAdvise( A->p, (anvec+1) * sizeof (int64_t), hipMemAdviseSetReadMostly, device) ;
    hipMemAdvise( A->i, anz * sizeof (int64_t), hipMemAdviseSetReadMostly, device) ;
    hipMemAdvise( A->x, anz * A->type->size, hipMemAdviseSetReadMostly,device) ;
    */
    hipMemPrefetchAsync( A->p, (anvec+1) * sizeof (int64_t), device, NULL); // stream_data) ;
    hipMemPrefetchAsync( A->i, anz * sizeof (int64_t), device, NULL ) ; //stream_data) ;
    hipMemPrefetchAsync( A->x, anz * A->type->size, device, NULL ) ; //stream_data) ;

    /*
    hipMemAdvise( B->p, (bnvec+1) * sizeof (int64_t), hipMemAdviseSetReadMostly, device) ;
    hipMemAdvise( B->i, bnz * sizeof (int64_t), hipMemAdviseSetReadMostly, device) ;
    hipMemAdvise( B->x, bnz * B->type->size, hipMemAdviseSetReadMostly, device) ;
    */
    hipMemPrefetchAsync( B->p, (bnvec+1) * sizeof (int64_t), device, NULL) ; //stream_data) ;
    hipMemPrefetchAsync( B->i, bnz * sizeof (int64_t), device, NULL ) ; //stream_data) ;
    hipMemPrefetchAsync( B->x, bnz * B->type->size, device, NULL ) ; //stream_data) ;

    

    // The work to compute C(i,j) is held in Ci [p], if C(i,j) appears in
    // as the pth entry in C.
    GB_callback mysemiring;
    const char *header_name = (const char *)"mySemiRing.h";
    mysemiring.load_string(header_name, semiring_code ) ;
    SR_callback_ptr = &mysemiring;


    //hipStream_t stream_AxB;
    //hipStreamCreate ( &stream_AxB);
    //----------------------------------------------------------------------
    // phase1: assign each C(i,j) to a bucket, and count them
    //----------------------------------------------------------------------
    dim3 grid( ntasks) ; 
    dim3 p2grid( (ntasks +  SYMBOLIC_PHASE_NTHREADS -1)
                          / (SYMBOLIC_PHASE_NTHREADS) ) ; 
    dim3 block( SYMBOLIC_PHASE_NTHREADS ) ;

    std::string base_name = "GB_jit_AxB_dot3_";
    std::string Opname = "phase1_" ;
    
    jitify::experimental::KernelLauncher phase1Kernel =
    jit::launcher( base_name + Opname + mask_name,
                   templates_GB_jit_AxB_dot3_phase1_cu,
                   header_names,
                   compiler_flags,
                   callback_wrapper) //,
                   //stream_AxB)
               .set_kernel_inst("GB_AxB_cuda_dot3_phase1",
                                {M->type->name})
               .configure(grid, block); 

    //----------------------------------------------------------------------
    // phase2: cumsum across the blockbuckets, propagate to thread level
    //----------------------------------------------------------------------
    base_name = "GB_jit_AxB_dot3_";
    Opname = "phase2";
    jitify::experimental::KernelLauncher phase2Kernel =
    jit::launcher( base_name + Opname,
                   templates_GB_jit_AxB_dot3_phase2_cu,
                   header_names,
                   compiler_flags,
                   callback_wrapper) //,
                   //stream_AxB)
               .set_kernel_inst("GB_AxB_dot3_phase2",
                                {})
               .configure(p2grid, block);

    base_name = "GB_jit_AxB_dot3_";
    Opname = "phase2";
    jitify::experimental::KernelLauncher phase2endKernel =
    jit::launcher( base_name + Opname,
                   templates_GB_jit_AxB_dot3_phase2_cu,
                   header_names,
                   compiler_flags,
                   callback_wrapper) //,
                   //stream_AxB)
               .set_kernel_inst("GB_AxB_dot3_phase2end",
                                {})
               .configure(grid, block);


    phase1Kernel.launch(
                        Nanobuckets,       // array of size NBUCKETS-blockDim.x-by-gridDim.x
                        Blockbucket,       // bucket counts, of size NBUCKETS-by-gridDim.x
                                           // input/output:
                        C,                 // final output matrix
                                           // inputs, not modified:
                        M,                 // mask matrix
                        A,                 // input matrix
                        B                  // input matrix
                    );


    // hipDeviceSynchronize();


    GBURBLE ("(GPU phase1 done) ") ;
    //for (int i = 0; i< cnz; i++){
    //  printf("C[%d] = %ld\n", i , Ci[i]);
    //}
    //----------------------------------------------------------------------
    // phase2: cumsum across the blockbuckets, propagate to thread level
    //----------------------------------------------------------------------
    int nblock = ntasks;

    phase2Kernel.launch(                    // input
                        Nanobuckets,       // array of size NBUCKETS-blockDim.x-by-gridDim.x
                        Blockbucket,       // bucket counts, of size NBUCKETS-by-gridDim.x
                                           // input/output:
                        Bucketp,           // global bucket cumsum, of size NBUCKETS+1
                        Bucket,            // global buckets, of size cnz (== mnz)
                        offset,
                        C,                 // final output matrix
                                           // inputs, not modified:
                        cnz,               // number of entries in mask and output matrix
                        nblock
                    );

    hipDeviceSynchronize();
    //hipMemPrefetchAsync( offset, (NBUCKETS) * sizeof (int64_t), hipCpuDeviceId, NULL) ;

    int64_t s= 0;
    for ( int bucket = 0 ; bucket < NBUCKETS+1; ++bucket)
    {
       Bucketp[bucket] = s; 
       s+= offset[bucket];
       //printf("bucketp[%d] = %ld\n", bucket, Bucketp[bucket]);
    }

    GBURBLE ("(GPU phase2 done) ") ;

    phase2endKernel.launch(                    // input
                        Nanobuckets,       // array of size NBUCKETS-blockDim.x-by-gridDim.x
                        Blockbucket,       // bucket counts, of size NBUCKETS-by-gridDim.x
                                           // input/output:
                        Bucketp,           // global bucket cumsum, of size NBUCKETS+1
                        Bucket,            // global buckets, of size cnz (== mnz)
                        offset,
                        C,                 // final output matrix
                                           // inputs, not modified:
                        cnz                // number of entries in mask and output matrix
                    );

    hipDeviceSynchronize();

    GBURBLE ("(GPU phase2end done) ") ;
    /* 
    for (int i = 0; i< cnz; i++){
      printf("C[%d],Bucket = %ld,%ld\n", i , Ci[i], Bucket[i]);
    }
    */
    
    //----------------------------------------------------------------------
    // phase3: do the numerical work
    //----------------------------------------------------------------------

    base_name = "GB_jit_";
    std::string kernel_name = "AxB_dot3_phase3_";
    C->nzombies = Bucketp[1];  //set pre-zombie counts

    for ( int bucket = 1 ; bucket < NBUCKETS; ++bucket)
    {
        std::string Opname = "";
        int sz = 0 ;

        const char*  jit_template;

        int64_t start = Bucketp[bucket];
        int64_t end = Bucketp[bucket+1];

        //if( (end-start>0)  && (start == Bucketp[1]) ) start = Bucketp[0]; //add in zombie slots

        int64_t Cnz = end- start;

        int gridsz, blocksz;

        //Nothing to do, next bucket
        if ( Cnz == 0 ) continue;

        GBURBLE ("\n\n(GPU phase3 bucket,bucketsize= %d,%ld) ",bucket,Cnz) ;

        switch (bucket)
        {

            //--------------------------------------------------------------
            // not a bucket ... bring out your dead:
            //--------------------------------------------------------------

            case GB_BUCKET_ZOMBIE : // C(i,j) is a zombie (not a bucket)
                break ;

            //--------------------------------------------------------------
            // CUDA kernel: dndn, handles a single bucket:
            //--------------------------------------------------------------

            // both A(:,i) and B(:,j) are dense
            case GB_BUCKET_DNDN :
                Opname = "dndn" ;
                jit_template = templates_GB_jit_AxB_dot3_phase3_dndn_cu;
                blocksz = 32;
                gridsz = ( Cnz -1 + blocksz)/blocksz;
                break ;

            //--------------------------------------------------------------
            // CUDA kernel: spdn, handles 4 buckets:
            //--------------------------------------------------------------

            // A(:,i) is dense and B(:,j) is very sparse (< 256 entries)
            case GB_BUCKET_DNVS :
            // A(:,i) is very sparse (< 256 entries) and B(:,j) is dense
            case GB_BUCKET_VSDN :
                sz = 64 ;
                Opname = "spdn" ;
                jit_template = templates_GB_jit_AxB_dot3_phase3_spdn_cu;
                blocksz = 32;
                gridsz = ( Cnz -1 + blocksz)/blocksz;
                break ;

            // A(:,i) is dense and B(:,j) is sparse (>= 256 entries)
            case GB_BUCKET_DNSP :
            // A(:,i) is sparse (>= 256 entries) and B(:,j) is dense
            case GB_BUCKET_SPDN :
                sz = 256 ;
                Opname = "spdn" ;
                jit_template = templates_GB_jit_AxB_dot3_phase3_spdn_cu;
                blocksz = 32;
                gridsz = ( Cnz -1 + blocksz)/blocksz;
                break ;

            //--------------------------------------------------------------
            // CUDA kernel: vssp, handles 1 bucket, uses binary search:
            //--------------------------------------------------------------

            // A(:,i) is very sparse compared to B(:,j), or visa versa
            case GB_BUCKET_VSSP :
                Opname = "vssp" ;
                jit_template = templates_GB_jit_AxB_dot3_phase3_vssp_cu;
                blocksz = 32;
                gridsz = ( Cnz -1 + blocksz)/blocksz;
                break ;

            //--------------------------------------------------------------
            // CUDA kernel: vsvs, handles 4 buckets:
            //--------------------------------------------------------------

            // let len = nnz (A (:,i) + nnz (B (:,j)), then:
            
            case GB_BUCKET_VSVS_256 : sz += 256-64 ;
            case GB_BUCKET_VSVS_64 :  sz += 64-16  ;
            case GB_BUCKET_VSVS_16 :  sz += 16-4   ;
            case GB_BUCKET_VSVS_4 :   sz += 4      ;
                Opname = "vsvs" ;
                jit_template = templates_GB_jit_AxB_dot3_phase3_vsvs_cu;
                blocksz = 1024;
                gridsz = GB_IMIN( 1024*number_of_sms, ( Cnz  + blocksz -1 )/blocksz);
                gridsz =  ( Cnz  + blocksz -1 )/blocksz;
                /*
                Opname = "warpix" ;
                jit_template = templates_GB_jit_AxB_dot3_phase3_warpix_cu;
                blocksz = 32;
                gridsz =  GB_IMIN( (mnvec+15)/16, 256*number_of_sms);
                */
                break ;
            
            //--------------------------------------------------------------
            // CUDA kernel: mp, use the merge-path method:
            //--------------------------------------------------------------

            case GB_BUCKET_MERGEPATH :
                Opname = "mp" ;
                jit_template = templates_GB_jit_AxB_dot3_phase3_mp_cu;
                blocksz = 32;
                gridsz = ( Cnz -1 + blocksz)/blocksz;
                break ;

            case GB_BUCKET_WARP_IX :   sz = 32      ;
                Opname = "warpix" ;
                jit_template = templates_GB_jit_AxB_dot3_phase3_warpix_cu;
                blocksz = 32;
                gridsz =  GB_IMIN( (mnvec+15)/16, 256*number_of_sms);
                break ;

            default:
                break ;
        }

        dim3 grid(gridsz);
        dim3 block(blocksz);

        std::cout<< "Kernel name =" <<Opname<<std::endl; 
        GBURBLE ("(GPU phase3 launch st,end=%ld,%ld nblocks,blocksize= %d,%d )\n",start,end,gridsz,blocksz) ;
        jit::launcher( base_name + kernel_name + Opname + "_" + semiring_name,
                       jit_template,
                       header_names,
                       compiler_flags,
                       callback_wrapper)
                   .set_kernel_inst(kernel_name + Opname,
                                    { ctype->name,
                                      A->type->name,
                                      B->type->name,
                                      semiring->multiply->xtype->name,
                                      semiring->multiply->ytype->name,
                                      semiring->multiply->ztype->name  })
                   .configure(grid, block) //if commented, use implicit 1D configure in launch
                   .launch(
                            start,   // input/output:
                            end, // global bucket cumsum, of size NBUCKETS+1
                            Bucket,            // global buckets, of size cnz (== mnz)
                            C,                 // final output matrix
                                               // inputs, not modified:
                            M,                 // Mi used for column index
                            A,                 // A matrix
                            B,                 // B matrix
                            sz                 // only used for sparse-sparse cases

                        );

        hipDeviceSynchronize();
    }
    GBURBLE ("(GPU phase3 done) ") ;
    
    std::string reduce_kernel_name = "reduceNonZombiesWarp";
    const char*  jit_template;
    #define red_blocksz 1024
    jit_template = templates_reduceNonZombiesWarp_cu;
    int num_reduce_blocks = GB_IMIN( 32*number_of_sms, (cnz + red_blocksz -1)/ red_blocksz  ) ;
    dim3 red_grid( num_reduce_blocks ) ; 
    dim3 red_block( red_blocksz ) ;

    int32_t *block_sum;
    //hipMallocManaged ((void**) &block_sum, (num_reduce_blocks)*sizeof(int32_t)) ;
    block_sum = (int32_t*)GB_cuda_malloc( (num_reduce_blocks)*sizeof(int32_t)) ;

    GBURBLE ("(GPU reduce launch nblocks,blocksize= %d,%d )\n", num_reduce_blocks, red_blocksz) ;
    jit::launcher( reduce_kernel_name + "_" + semiring_name,
                   jit_template,
                   header_names,
                   compiler_flags,
                   callback_wrapper)
                   .set_kernel_inst( reduce_kernel_name , { ctype->name })
                   .configure(red_grid, red_block) //if commented, use implicit 1D configure in launch
                   .launch(
                            C->i,   // index vector, only sum up values >= 0
                            C->x,   // input pointer to vector to reduce, with zombies
                            block_sum,             // Block sums on return 
                            (unsigned int)cnz      // length of vector to reduce to scalar

                        );

    hipDeviceSynchronize();

    int32_t num_triangles = 0;
    for (int i = 0; i< num_reduce_blocks; i++){
       //printf("block%d num_triangles = %d\n", i, block_sum[i] );
       num_triangles += block_sum[i] ;
    }
    printf("num_triangles = %d\n",  num_triangles );

    GB_cuda_free( block_sum ); 
    //hipMemPrefetchAsync( C->p, (mnvec+1) * sizeof (int64_t), hipCpuDeviceId, NULL) ; //stream_data ) ;
    //hipMemPrefetchAsync( C->i, cnz * sizeof (int64_t), hipCpuDeviceId, NULL ) ; //stream_data ) ;
    //hipMemPrefetchAsync( C->x, cnz * sizeof (int32_t), hipCpuDeviceId, NULL ) ; //stream_data ) ;
    /*
    hipMemcpy( Citemp, C->i, cnz * sizeof( int64_t), hipMemcpyDefault );    
    hipMemcpy( Cxtemp, C->x, cnz * C->type->size, hipMemcpyDefault );    
    GB_cuda_free( C->i);
    GB_cuda_free( C->x);
    C->i = Citemp;
    C->x = Cxtemp;
    */

    hipDeviceSynchronize();

    return GrB_SUCCESS; 
}

